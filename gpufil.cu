#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <string>
#include <utility>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include "dada.hpp"
#include "errors.hpp"
#include "filterbank.hpp"

using std::cerr;
using std::cout;
using std::endl;
using std::ifstream;
using std::ofstream;
using std::string;
using std::vector;

#define DEBUG 0
#define GPURUN 1
#define NACCUMULATE 4000
#define PERBLOCK 625
#define TIMESCALE 0.125
#define UNPACKFACTOR 4
#define VDIFSIZE 8000
#define FFTOUT 257
#define FFTUSE 256

#define COMPLEX 1
#define INCHANS 1
#define OUTCHANS 1024
#define NPOL 2
#define TIMEAVG 4
#define INBANDS 1

struct FrameInfo {
    unsigned int frameno;
    unsigned int refsecond;
    unsigned int refepoch;
};

struct Timing {
    float readtime;
    float scaletime;
    float filtime;
    float savetime;
    float totaltime;
    float intertime;
};
 /*
struct FactorFunctor {
    __host__ __device__ float operator()(float val) {
        return val != 0 ? 1.0f/val : val;
    }
};

__constant__ unsigned char kMask[] = {0x03, 0x0C, 0x30, 0xC0};

__global__ void UnpackKernel(unsigned char **in, float **out, size_t samples) {

    // NOTE: Each thread in the block processes 625 incoming bytes
    int idx = blockIdx.x * blockDim.x * PERBLOCK + threadIdx.x;
    int tmod = threadIdx.x % 4;

    // NOTE: Each thread can store one value
    __shared__ unsigned char incoming[1024];

    int outidx = blockIdx.x * blockDim.x * PERBLOCK * 4;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {
        if (idx < samples) {
            for (int ipol = 0; ipol < 2; ++ipol) {
                incoming[threadIdx.x] = in[ipol][idx];
                __syncthreads();
                int outidx2 = outidx + threadIdx.x;
		for (int ichunk = 0; ichunk < 4; ++ichunk) {
                    int inidx = threadIdx.x / 4 + ichunk * 256;
                    unsigned char inval = incoming[inidx];
                    out[ipol][outidx2] = static_cast<float>(static_cast<short>(((inval & kMask[tmod]) >> (2 * tmod))));
                    outidx2 += 1024;
                }
            }
        }
        idx += blockDim.x;
        outidx += blockDim.x * 4;
    }
}

// NOTE: Does not do any frequency averaging
// NOTE: Outputs only the total intensity and no other Stokes parameters
// NOTE: PERBLOCK is the number of output samples per block
__global__ void DetectKernel(hipfftComplex** __restrict__ in, float* __restrict__ out) {
    int outidx = blockIdx.x * PERBLOCK * FFTUSE + FFTUSE - threadIdx.x - 1;
    int inidx = blockIdx.x * PERBLOCK * TIMEAVG * FFTOUT + threadIdx.x + 1;

    float outvalue = 0.0f;
    hipfftComplex polval;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {

        // NOTE: Read the data from the incoming array
        for (int ipol = 0; ipol < 2; ++ipol) {
            for (int iavg = 0; iavg < TIMEAVG; ++iavg) {
                polval = in[ipol][inidx + iavg * FFTOUT];
                outvalue += polval.x * polval.x + polval.y * polval.y;
            }

        }
        outvalue *= TIMESCALE;
        out[outidx] = outvalue;
        inidx += FFTOUT * TIMEAVG;
        outidx += FFTUSE;
        outvalue = 0.0;
    }
}

__global__ void DetectScaleKernel(hipfftComplex** __restrict__ in, unsigned char* __restrict__ out, float* __restrict__ means, float* __restrict__ stdevs) {
    int outidx = blockIdx.x * PERBLOCK * FFTUSE + FFTUSE - threadIdx.x - 1;
    int inidx = blockIdx.x * PERBLOCK * TIMEAVG * FFTOUT + threadIdx.x + 1;

    float outvalue = 0.0f;
    hipfftComplex polval;

    int scaled = 0;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {

        // NOTE: Read the data from the incoming array
        for (int ipol = 0; ipol < 2; ++ipol) {
            for (int iavg = 0; iavg < TIMEAVG; ++iavg) {
                polval = in[ipol][inidx + iavg * FFTOUT];
                outvalue += polval.x * polval.x + polval.y * polval.y;
            }

        }
        outvalue *= TIMESCALE;
        scaled = __float2int_ru((outvalue - means[FFTUSE - threadIdx.x - 1]) / stdevs[FFTUSE - threadIdx.x - 1] * 32.0f + 128.0f);
        if (scaled > 255) {
            scaled = 255;
        } else if (scaled < 0) {
            scaled = 0;
        }
        out[outidx] = (unsigned char)scaled;
        inidx += FFTOUT * TIMEAVG;
        outidx += FFTUSE;
        outvalue = 0.0;
    }
}

__global__ void InitDivFactors(float *factors, size_t togenerate) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // NOTE: I don't want to be dividing by 0
    // NOTE: idx of 0 will not be used anyway
    if (idx < togenerate) {
        if (idx != 0) {
            factors[idx] = 1.0f / idx;
        } else {
            factors[idx] = idx;
        }
    }
}

__global__ void GetScalingFactorsKernel(float* __restrict__ indata, float *base, float *stdev, float *factors, int processed) {

    // NOTE: Filterbank file format coming in
    //float mean = indata[threadIdx.x];
    float mean = 0.0f;
    // NOTE: Depending whether I save STD or VAR at the end of every run
    // float estd = stdev[threadIdx.x];
    float estd = stdev[threadIdx.x] * stdev[threadIdx.x] * (processed - 1.0f);
    float oldmean = base[threadIdx.x];

    //float estd = 0.0f;
    //float oldmean = 0.0;

    float val = 0.0f;
    float diff = 0.0;
    // NOTE: There are 15625 output time samples per NACCUMULATE frames
    for (int isamp = 0; isamp < 15625; ++isamp) {
        val = indata[isamp * FFTUSE + threadIdx.x];
        diff = val - oldmean;
        mean = oldmean + diff * factors[processed + isamp + 1];
        estd += diff * (val - mean);
        oldmean = mean;
    }
    base[threadIdx.x] = mean;
    stdev[threadIdx.x] = sqrtf(estd / (float)(processed + 15625 - 1.0f));
    // stdev[threadIdx.x] = estd;
}
*/

// NOTE: Not really optimised yet
__global__ void UnpackDadaKernel(int ntimes, uchar4* __restrict__ indata, hipfftComplex* __restrict__ outdata) {

    uchar4 tmpread;

    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < ntimes; idx += gridDim.x * blockDim.x) {
        
        // tmpread = indata[idx];
        // outdata[idx].x = (float)(tmpread.x);
        // outdata[idx].y = (float)(tmpread.y);
        // outdata[idx + ntimes].x = (float)(tmpread.z);
        // outdata[idx + ntimes].y = (float)(tmpread.w);

        tmpread = indata[idx];
        outdata[idx].x = static_cast<float>(static_cast<unsigned short>(tmpread.x));
        outdata[idx].y = static_cast<float>(static_cast<unsigned short>(tmpread.y));
        outdata[idx + ntimes].x = static_cast<float>(static_cast<unsigned short>(tmpread.z));
        outdata[idx + ntimes].y = static_cast<float>(static_cast<unsigned short>(tmpread.w));

    }

}

__global__ void DetectDadaKernel(int ntimes, hipfftComplex* __restrict__ fftdata, float* __restrict__ powerdata) {

    //int inidx;
    //int outidx;
    int timeoffset;
    int poloffset = ntimes * OUTCHANS;

    float power = 0.0;;
    hipfftComplex tmpvalue;
    for (int timeidx = blockIdx.x * TIMEAVG; timeidx < ntimes; timeidx += gridDim.x * TIMEAVG) {

        timeoffset = timeidx * OUTCHANS;

        for (int iavg = 0; iavg < TIMEAVG; ++iavg) {
            tmpvalue = fftdata[timeoffset + iavg * OUTCHANS + threadIdx.x];
            power += tmpvalue.x * tmpvalue.x + tmpvalue.y * tmpvalue.y;
            tmpvalue = fftdata[poloffset + timeoffset + iavg * OUTCHANS + threadIdx.x];
            power += tmpvalue.x * tmpvalue.x + tmpvalue.y * tmpvalue.y;
        }

        powerdata[timeoffset / TIMEAVG + threadIdx.x] = power;
        power = 0.0f;
    }

}

// NOTE: This is a very naive approach, but it works fast enough for now
__global__ void BandpassKernel(int ntimes, float* __restrict__ powerdata, float* __restrict__ bandpass) {

    float sum;

    sum = 0.0f;

    for (int isamp = 0; isamp < ntimes; ++isamp) {
        sum += powerdata[isamp * OUTCHANS + threadIdx.x];
    }

    bandpass[threadIdx.x] += sum;

}

int main(int argc, char *argv[]) {

    string inpola = "";
    string inpolb = "";
    string outfil = "";
    string config = "";
    string dadastr = "";
    double readsec; 
    bool scaling = false;
    bool saveinter = false;

    if ((argc < 5) || (argv[1] == "-h") || (argv[1] == "--help")) {
        cout << "Incorrect number of arguments!" << endl;
        cout << "Command line options:" << endl
                << "-a <filename> - input file for polarisation a" << endl
                << "-b <filename> - input file for polarisation b" << endl
                << "-d <filename> - input DADA file" << endl
                << "-o <filename> - output filterbank file" << endl
                << "-c <filename> - input configuration file" << endl
                << "-r <number> - number of seconds to process - CURRENTLY NOT WORKING" << endl
                << "-s - enable scaling from 32 bits to 8 bits" << endl
                << "-i - enable saving the intermediate data products" << endl
                << "-h, --help - display this message" << endl;
        exit(EXIT_SUCCESS);
    }

    for (int iarg = 0; iarg < argc; ++iarg) {
        if (std::string(argv[iarg]) == "-a") {
            iarg++;
            inpola = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-b") {
            iarg++;
            inpolb = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-d") {
            iarg++;
            dadastr = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-o") {
            iarg++;
            outfil = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-c") {
            iarg++;
            config = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-s") {
            cout << "Will scale the data to 8 bits" << endl;
            scaling = true;
        } else if (std::string(argv[iarg]) == "-i") {
            cout << "Will save the intermediate products" << endl;
            saveinter = true;
        } else if (std::string(argv[iarg]) == "-r") {
            iarg++;
            readsec = std::stod(argv[iarg]);
        }
    }

    if (!inpola.empty() && !dadastr.empty()) {
        cerr << "It's one or the other: DADA or VDIF, not both!" << endl;
        return 1;
    }

    if (!dadastr.empty()) {

        cout << "Input file: " << dadastr << endl;
        
        std::ifstream indada(dadastr.c_str(), std::ios_base::binary);        
        long long filesize = 0;
        indada.seekg(0, indada.end);
        filesize = indada.tellg() - 4096L;
        indada.seekg(0, indada.beg);

        // NOTE: 4 bytes per full time sample: 1 byte sampling, 2 polarisations, complex number
        size_t totalsamples = filesize / 4;
        if (filesize != totalsamples * 4) {
            std::cerr << "A non-integer number of time samples was read - something went very wrong!" << std::endl;
            return 1;
        }

        std::cout << "File size: " << filesize / 1024.0f / 1024.0f << "MiB with " << totalsamples << " time samples" << std::endl;
        // NOTE: That simply ensures that we only process the integer number of final filterbank channels
        totalsamples = (int)((float)totalsamples / (OUTCHANS * TIMEAVG)) * OUTCHANS * TIMEAVG;
        std::cout << "Will use first" << totalsamples << " samples" << std::endl;

        FilHead filhead = {};
        ReadDadaHeader(indada, filhead);
        if (!scaling) {
            filhead.nbits = 32;
        }
        filhead.nchans = OUTCHANS;
        filhead.tsamp = filhead.tsamp * OUTCHANS * TIMEAVG;
        PrintFilterbankHeader(filhead);
        
        size_t freemem = 0;
        size_t totalmem = 0;
        cudaCheckError(hipMemGetInfo(&freemem, &totalmem));
        // NOTE: Let's liffh just 25% of what's free, because cuFFT happens...
        freemem = freemem * 0.25;
        std::cout << "Total memory: " << totalmem / 1024.0f / 1024.0f << "MiB, with " << freemem / 1024.0f / 1024.0f << "MiB free" << std::endl;
        
        // original file + original file cast to hipfftComplex for FFT + output filterbank file saved as 32 bit float
        size_t needmem = 4 * totalsamples + 4 * totalsamples * 4 + totalsamples / OUTCHANS / TIMEAVG * OUTCHANS * 4;
        std::cout << "Need " << needmem / 1024.0f / 1024.0f << "MiB on the device" << std::endl;
        
        int nblocks = 0;
        size_t sampperblock = 0;
        size_t remsamp = 0;
        
        if (needmem < freemem) {
            std::cout << "Can store everything in global memory at once..." << std::endl;
            nblocks = 1;
            sampperblock = totalsamples;
        } else {
            std::cout << "We need to divide the job..." << std::endl;

            sampperblock = (int)((float)freemem / ((float)(OUTCHANS * TIMEAVG) * (4.0f + 16.0f + 4.0f / (float)TIMEAVG))) * OUTCHANS * TIMEAVG;
            nblocks = (int)(totalsamples / sampperblock);
            remsamp = totalsamples - nblocks * sampperblock;

            std::cout << "Will process the data in " << nblocks << " blocks, with "
                        << sampperblock << " samples per block";
            if (remsamp) {
                std::cout << " and an extra block with " << remsamp << " samples at the end";
            }
            std::cout << std::endl;
        }

        std::ofstream filfile(outfil.c_str(), std::ios_base::binary);
        WriteFilterbankHeader(filfile, filhead);

        /**** ####
        // STAGE: MEMORY AND FFT
        #### ****/
        // NOTE: Factor of 4 to account for 2 polarisations and complex components for every time sample
        size_t blockread = sampperblock * 4;
        size_t remread = remsamp * 4;
        
        hipfftHandle fftplan;
        int fftsizes[1];
        fftsizes[0] = OUTCHANS;
        // NOTE: Factor of 2 to account for 2 polarisations
        int fftbatchsize = sampperblock * 2 / fftsizes[0];
        cufftCheckError(hipfftPlanMany(&fftplan, 1, fftsizes, NULL, 1, OUTCHANS, NULL, 1, OUTCHANS, HIPFFT_C2C, fftbatchsize));
 
        unsigned char *hostvoltage = new unsigned char[blockread];
        unsigned char *devicevoltage = new unsigned char[blockread];
        cudaCheckError(hipMalloc((void**)&devicevoltage, blockread * sizeof(unsigned char)));

        hipfftComplex *devicefft;
        cudaCheckError(hipMalloc((void**)&devicefft, sampperblock * 2 * sizeof(hipfftComplex)));

        size_t powersize = sampperblock / OUTCHANS * OUTCHANS / TIMEAVG;
        float *hostpower = new float[powersize];
        float *devicepower;
        cudaCheckError(hipMalloc((void**)&devicepower, powersize * sizeof(float)))

        float *hostband = new float[OUTCHANS];
        float *deviceband;
        cudaCheckError(hipMalloc((void**)&deviceband, OUTCHANS * sizeof(float)));

        // NOTE: Just in case I did something wrong
        indada.seekg(4096, indada.beg);

        for (int iblock = 0; iblock < nblocks; iblock++) {

            std::cout << "Processing block " << iblock << "..." << std::endl;

            indada.read(reinterpret_cast<char*>(hostvoltage), blockread * sizeof(unsigned char));

            cudaCheckError(hipMemcpy(devicevoltage, hostvoltage, blockread * sizeof(unsigned char), hipMemcpyHostToDevice));

            dim3 block (OUTCHANS, 1, 1);
            dim3 grid (64, 1, 1);

            UnpackDadaKernel<<<grid, block, 0, 0>>>(sampperblock, reinterpret_cast<uchar4*>(devicevoltage), devicefft);
            cudaCheckError(hipGetLastError());

            cufftCheckError(hipfftExecC2C(fftplan, devicefft, devicefft, HIPFFT_FORWARD));

            DetectDadaKernel<<<grid, block, 0, 0>>>(sampperblock / OUTCHANS, devicefft, devicepower);
            cudaCheckError(hipGetLastError());

            BandpassKernel<<<1, OUTCHANS, 0, 0>>>(sampperblock / OUTCHANS / TIMEAVG, devicepower, deviceband);
            cudaCheckError(hipGetLastError());

            cudaCheckError(hipMemcpy(hostpower, devicepower, powersize * sizeof(float), hipMemcpyDeviceToHost));

            filfile.write(reinterpret_cast<char*>(hostpower), powersize * sizeof(float));
        } 
        
        cufftCheckError(hipfftDestroy(fftplan));

        if (remsamp) {

            std::cout << "Processing the remainder block..." << std::endl;

            indada.read(reinterpret_cast<char*>(hostvoltage), remread * sizeof(unsigned char));

            cudaCheckError(hipMemcpy(devicevoltage, hostvoltage, remread * sizeof(unsigned char), hipMemcpyHostToDevice));

            dim3 block(OUTCHANS, 1, 1);
            dim3 grid (64, 1, 1);

            UnpackDadaKernel<<<grid, block, 0, 0>>>(remsamp, reinterpret_cast<uchar4*>(devicevoltage), devicefft);
            cudaCheckError(hipGetLastError());

            hipfftHandle fftplanrem;
            int fftrembatchsize = remsamp * 2 / fftsizes[0];
            cufftCheckError(hipfftPlanMany(&fftplanrem, 1, fftsizes, NULL, 1, OUTCHANS, NULL, 1, OUTCHANS, HIPFFT_C2C, fftrembatchsize));

            cufftCheckError(hipfftExecC2C(fftplanrem, devicefft, devicefft, HIPFFT_FORWARD));

            DetectDadaKernel<<<grid, block, 0, 0>>>(remsamp / OUTCHANS, devicefft, devicepower);
            cudaCheckError(hipGetLastError());

            BandpassKernel<<<1, OUTCHANS, 0, 0>>>(remsamp / OUTCHANS / TIMEAVG, devicepower, deviceband);
            cudaCheckError(hipGetLastError());

            cudaCheckError(hipMemcpy(hostpower, devicepower, remsamp / OUTCHANS / TIMEAVG * OUTCHANS * sizeof(float), hipMemcpyDeviceToHost));

            filfile.write(reinterpret_cast<char*>(hostpower), remsamp / OUTCHANS / TIMEAVG * OUTCHANS * sizeof(float));

            cufftCheckError(hipfftDestroy(fftplanrem)); 
        }

        cudaCheckError(hipMemcpy(hostband, deviceband, OUTCHANS * sizeof(float), hipMemcpyDeviceToHost));

        std::ofstream bandout("bandpass.dat");

        if (bandout) {
            for (int ichan = 0; ichan < OUTCHANS; ++ichan) {
                bandout << hostband[ichan] << std::endl;
            }
        }

        bandout.close();
        filfile.close();
        indada.close();

        hipFree(deviceband);
        hipFree(devicepower);
        hipFree(devicefft);
        hipFree(devicevoltage);

        delete [] hostband;
        delete [] hostpower;
        delete [] hostvoltage;

    } 

    // NOTE: This code is in a bit of a bad state - need to clean it up
    // else if (!inpola.empty() & !inpolb.empty()) {

    //     cout << "Input files: " << inpola << " " << inpolb << endl;

    //     FilHead filhead;
    //     ReadFilterbankHeader(config, filhead);



        
    //     if (scaling) {
    //         filhead.nbits = 8;
    //     }
    
    //     // TODO: This will be wrong for R2C FFT
    //     filhead.tsamp = 1.0 / (2.0 * filhead.foff) * 2 * FFTUSE * TIMEAVG;
    //     // TODO: Make sure it is the middle of the top frequency channel
    //     filhead.fch1 = (filhead.fch1 + filhead.foff / 2.0f) * 1e-06;
    //     filhead.nchans = FFTUSE;
    //     filhead.foff = -1.0 * filhead.foff / FFTUSE * 1e-06 ;
    
    //     filhead.fch1 = filhead.fch1 + filhead.foff / 2.0;
    
    //     if (DEBUG) {
    //         cout << "Some header info:\n"
    //                 << "Raw file: " << filhead.rawfile << endl
    //                 << "Source name: " << filhead.source << endl
    //                 << "Azimuth: " << filhead.az << endl
    //                 << "Zenith angle: " << filhead.za << endl
    //                 << "Declination: " << filhead.dec << endl
    //                 << "Right ascension: " << filhead.ra << endl
    //                 << "Top channel frequency: " << filhead.fch1 << endl
    //                 << "Channel bandwidth: " << filhead.foff << endl
    //                 << "Number of channels: " << filhead.nchans << endl
    //                 << "Sampling time: " << filhead.tsamp << endl
    //                 << "Bits per sample: " << filhead.nbits << endl;
    //     }
    
    //     // TODO: Make sure there are correct values for bandwidth and sampling time in the header after taking averaging into account
    
    //     ifstream filepola(inpola.c_str(), ifstream::in | ifstream::binary);
    //     ifstream filepolb(inpolb.c_str(), ifstream::in | ifstream::binary);
    //     ofstream filfile(outfil.c_str(), ofstream::out | ofstream::binary);
    
    //     if (!filepola || !filepolb) {
    //     if (!filepola) {
    //             cout << "Could not open file " << inpola << endl;
    //         }
    //         if (!filepolb) {
    //             cout << "Could not open file " << inpolb << endl;
    //         }
    //         exit(EXIT_FAILURE);
    //     }
    //     // TODO: Can save the filterbank header straight away, after the first header is read
    //     unsigned char vdifheadpola[32];
    //     unsigned char vdifheadpolb[32];
    //     filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //     filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    
    //     filepola.seekg(0, filepola.end);
    //     long long filelengtha = filepola.tellg();
    //     filepola.seekg(0, filepola.beg);
    
    //     filepolb.seekg(0, filepolb.end);
    //     long long filelengthb = filepolb.tellg();
    //     filepolb.seekg(0, filepolb.beg);
    
    //     unsigned int startframe;
    //     unsigned int startsecond;
    
    //     cout << filelengtha << " " << filelengthb << endl;
    
    //     startframe = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));	// frame number in this second
    //     startsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    
    //     if (DEBUG) {
    //         cout << "Starting time: " << startsecond << ":" << startframe << endl;
    //     }
    
    //     // NOTE: Need to read headers in
    //     unsigned int toread = NACCUMULATE * 8000;
    //     // NOTE: No more headers after unpacking
    //     unsigned int unpackedsize = NACCUMULATE * VDIFSIZE * UNPACKFACTOR;
    //     unsigned int fftedsize = unpackedsize / (2 * FFTUSE) * FFTOUT;
    //     unsigned int powersize = unpackedsize / (2 * FFTUSE) * FFTUSE / TIMEAVG;
    
    //     hipfftHandle fftplan;
    //     int fftsizes[1];
    //     fftsizes[0] = 2 * FFTUSE;
    //     int fftbatchsize = unpackedsize / fftsizes[0];
    //     cout << fftbatchsize << endl;
    //     cufftCheckError(hipfftPlanMany(&fftplan, 1, fftsizes, NULL, 1, FFTUSE, NULL, 1, FFTUSE, HIPFFT_R2C, fftbatchsize));
    
    //     unsigned char *tmppola = new unsigned char[toread];
    //     unsigned char *tmppolb = new unsigned char[toread];
    
    //     unsigned char *devpola;
    //     unsigned char *devpolb;
    //     unsigned char **datapol = new unsigned char*[NPOL];
    //     unsigned char **devpol;
    //     float **unpacked = new float*[NPOL];
    //     float **devunpacked;
    //     hipfftComplex **ffted = new hipfftComplex*[NPOL];
    //     hipfftComplex **devffted;
        
    //     unsigned char *devpower;
    //     unsigned char *tmppower = new unsigned char[powersize * filhead.nbits / 8];
    
    //     if (GPURUN) {
    //         cudaCheckError(hipMalloc((void**)&devpola, toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&devpolb, toread * sizeof(unsigned char)));
    
    //         cudaCheckError(hipMalloc((void**)&devpol, NPOL * sizeof(unsigned char*)));
    //         cudaCheckError(hipMalloc((void**)&datapol[0], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&datapol[1], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMemcpy(devpol, datapol, NPOL * sizeof(unsigned char*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devunpacked, NPOL * sizeof(float*)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[0], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[1], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMemcpy(devunpacked, unpacked, NPOL * sizeof(float*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devffted, NPOL * sizeof(hipfftComplex*)));
    //         cudaCheckError(hipMalloc((void**)&ffted[0], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMalloc((void**)&ffted[1], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMemcpy(devffted, ffted, NPOL * sizeof(hipfftComplex*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devpower, powersize * (filhead.nbits / 8)));
    //     }
    
    //     vector<std::pair<FrameInfo, FrameInfo>> vdifframes;
    
    //     FrameInfo tmpframea, tmpframeb;
    //     int refsecond;
    //     int frameno;
    //     int epoch;
    
    //     WriteFilterbankHeader(filfile, filhead);
       
    //     Timing runtimes;
    //     runtimes.readtime = 0.0f;
    //     runtimes.scaletime = 0.0f;
    //     runtimes.filtime = 0.0f;
    //     runtimes.savetime = 0.0f;
    //     runtimes.totaltime = 0.0f;
    //     runtimes.intertime = 0.0f;
    
    //     std::chrono::time_point<std::chrono::steady_clock> readstart, readend, scalestart, scaleend, filstart, filend, savestart, saveend, interstart, interend;
    
    //     float *tmpunpackeda = new float[unpackedsize];
    //     float *tmpunpackedb = new float[unpackedsize];
    //     hipfftComplex *tmpffta = new hipfftComplex[fftedsize];
    //     hipfftComplex *tmpfftb = new hipfftComplex[fftedsize];
    
    //     bool saved = false;
    
    //     //float *dmeans;
    //     //float *dstdevs;
    //     //cudaCheckError(hipMalloc((void**)&dmeans, FFTUSE * sizeof(float)));
    //     //cudaCheckError(hipMalloc((void**)&dstdevs, FFTUSE * sizeof(float)));
    
    //     thrust::device_vector<float> dmeans, dstdevs;
    //     dmeans.resize(FFTUSE);
    //     dstdevs.resize(FFTUSE);
    //     thrust::fill(dmeans.begin(), dmeans.end(), 0.0f);
    //     thrust::fill(dstdevs.begin(), dstdevs.end(), 0.0f);
    //     float *pdmeans = thrust::raw_pointer_cast(dmeans.data());
    //     float *pdstdevs = thrust::raw_pointer_cast(dstdevs.data());    
    
    //     cout << "Size of the device vectors: " << dmeans.size() << " " << dstdevs.size() << endl;
    
    //     scalestart = std::chrono::steady_clock::now();
    
    //     // NOTE: Use first 5 accumulates of data to obtain scaling factors
    //     if (scaling) {
    
    //         size_t divfactors = 5 * powersize / FFTUSE;
    //         thrust::device_vector<float> dfactors; 
    //         dfactors.resize(divfactors + 1);
    //         thrust::sequence(dfactors.begin(), dfactors.end());
    //         thrust::transform(dfactors.begin(), dfactors.end(), dfactors.begin(), FactorFunctor());
    //         float *pdfactors = thrust::raw_pointer_cast(dfactors.data());
    
    //         //float *dfactors;
    //         //size_t divfactors = 5 * powersize / FFTUSE;
    //         //cudaCheckError(hipMalloc((void**)&dfactors, divfactors * sizeof(float)));
    //         //int scalethreads = 1024;
    //         //int scaleblocks = (divfactors - 1) / scalethreads + 1;
    //         //cout << "Div factors blocks: " << scaleblocks << " and threads: " << scalethreads << endl;
    //         //InitDivFactors<<<scaleblocks, scalethreads>>>(dfactors, divfactors);
    //         //cudaCheckError(hipDeviceSynchronize());
    //         //cudaCheckError(hipGetLastError());
    //         size_t processed = 0;
    
    //         float *tmpdpower;
    //         cudaCheckError(hipMalloc((void**)&tmpdpower, powersize * sizeof(float)));
    
    //     while((filepola.tellg() < (5 * NACCUMULATE * 8032)) && (filepolb.tellg() < (5 * NACCUMULATE * 8032))) {
    //             for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //                 filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //                 filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //                 filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //                 filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             }
    
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    //             DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, tmpdpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             GetScalingFactorsKernel<<<1, FFTUSE, 0, 0>>>(tmpdpower, pdmeans, pdstdevs, pdfactors, processed);
    //             processed += (powersize / FFTUSE);
    //             cudaCheckError(hipDeviceSynchronize());
    //         }
    
    //         //float *hmeans = new float[FFTUSE];
    //         //float *hstdevs = new float[FFTUSE];
    
    //         //cudaCheckError(hipMemcpy(hmeans, dmeans, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    //         //cudaCheckError(hipMemcpy(hstdevs, dstdevs, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    
    //         thrust::host_vector<float> hmeans = dmeans;
    //         thrust::host_vector<float> hstdevs = dstdevs;
    
    //         std::ofstream statsfile("mean_stdev.dat");
    
    //         cout << "Size of host vector:" << hmeans.size() << endl;
     
    //         if (statsfile) {
    //             for (int ichan = 0; ichan < hmeans.size(); ++ichan) {
    //                 statsfile << hmeans[ichan] << " " << hstdevs[ichan] << endl;
    //             }
    //         } else {
    //             cerr << "Could not open the stats file" << endl;
    //         }
    
    //         statsfile.close();
    
    //         hipFree(tmpdpower);
             
    //     }
    
    //     scaleend = std::chrono::steady_clock::now();
    
    //     runtimes.scaletime = std::chrono::duration<float>(scaleend - scalestart).count();
    
    //     filepola.seekg(0, filepola.beg);
    //     filepolb.seekg(0, filepolb.beg);
    
    //     std::ofstream unpackedfilea ((outfil + ".unp0").c_str(), std::ios_base::binary);
    //     std::ofstream unpackedfileb ((outfil + ".unp1").c_str(), std::ios_base::binary);
    //     std::ofstream fftfilea ((outfil + ".fft0").c_str(), std::ios_base::binary);
    //     std::ofstream fftfileb ((outfil + ".fft1").c_str(), std::ios_base::binary);
    
    //     while((filepola.tellg() < (filelengtha - NACCUMULATE * 8000)) && (filepolb.tellg() < (filelengthb - NACCUMULATE * 8000))) {
    //         //cout << filepola.tellg() << endl;
    //         // NOTE: This implementation
    //         for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //         readstart = std::chrono::steady_clock::now();
    //             filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //             filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //             filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //             filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             readend = std::chrono::steady_clock::now();
    //             runtimes.readtime += std::chrono::duration<float>(readend - readstart).count();
    
    //             refsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));
    //             epoch = (unsigned int)(vdifheadpola[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframea.frameno = frameno;
    //             tmpframea.refsecond = refsecond;
    //             tmpframea.refepoch = epoch;
    
    //             refsecond = (unsigned int)(vdifheadpolb[0] | (vdifheadpolb[1] << 8) | (vdifheadpolb[2] << 16) | ((vdifheadpolb[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpolb[4] | (vdifheadpolb[5] << 8) | (vdifheadpolb[6] << 16));
    //             epoch = (unsigned int)(vdifheadpolb[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframeb.frameno = frameno;
    //             tmpframeb.refsecond = refsecond;
    //             tmpframeb.refepoch = epoch;
    
    //             vdifframes.push_back(std::make_pair(tmpframea, tmpframeb));
    
    //             // NOTE: Can use subtract startframe to put frame count at 0 and use that to save into the buffer
    
    //         }
     
    //        if (GPURUN) {
    //             filstart = std::chrono::steady_clock::now();
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    
    //             if (filhead.nbits == 8) {
    //                 DetectScaleKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<unsigned char*>(devpower), pdmeans, pdstdevs);
    //             } else if (filhead.nbits == 32) {
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             } else {
    //                 cerr << "Unsupported option! Will use float!" << endl;
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             }
    
    //             //PowerKernel<<<25, FFTUSE, 0, 0>>>(devffted, devpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             cudaCheckError(hipMemcpy(tmppower, devpower, powersize * filhead.nbits / 8, hipMemcpyDeviceToHost));
                
    //             if (!saved) {
    //                 std::ofstream unpackedfile("unpacked.dat");
    //         cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         for (int isamp = 0; isamp < 2 * 8000 * 4; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }
    //                 unpackedfile.close();
    //                 saved = true;
    //             }
    
    //             filend = std::chrono::steady_clock::now();
    //             runtimes.filtime += std::chrono::duration<float>(filend - filstart).count();
                
    //             savestart = std::chrono::steady_clock::now(); 
    //             filfile.write(reinterpret_cast<char*>(tmppower), powersize * filhead.nbits / 8);
    //             saveend = std::chrono::steady_clock::now();
    //             runtimes.savetime += std::chrono::duration<float>(saveend - savestart).count();
    
       
    
    //             if (saveinter) {
    
    //                 interstart = std::chrono::steady_clock::now();
    
    //                 cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < unpackedsize; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }*/
    
    //                 unpackedfilea.write(reinterpret_cast<char*>(tmpunpackeda), unpackedsize * sizeof(float));
    //                 unpackedfileb.write(reinterpret_cast<char*>(tmpunpackedb), unpackedsize * sizeof(float));
    
    //                 cudaCheckError(hipMemcpy(tmpffta, ffted[0], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 cudaCheckError(hipMemcpy(tmpfftb, ffted[1], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < fftedsize; ++isamp) {
    //                     fftfile << tmpffta[isamp].x << " " << tmpffta[isamp].y << " " << tmpfftb[isamp].x << " " << tmpfftb[isamp].y << endl;
    //                 }*/
    
    //                 fftfilea.write(reinterpret_cast<char*>(tmpffta), fftedsize * sizeof(hipfftComplex));
    //                 fftfileb.write(reinterpret_cast<char*>(tmpfftb), fftedsize * sizeof(hipfftComplex));
    
    //                 interend = std::chrono::steady_clock::now();
    //                 runtimes.intertime += std::chrono::duration<float>(interend - interstart).count();           
    
    //             }FilHead filhead;
    //     ReadFilterbankHeader(config, filhead);
        
    //     if (scaling) {
    //         filhead.nbits = 8;
    //     }
    
    //     // TODO: This will be wrong for R2C FFT
    //     filhead.tsamp = 1.0 / (2.0 * filhead.foff) * 2 * FFTUSE * TIMEAVG;
    //     // TODO: Make sure it is the middle of the top frequency channel
    //     filhead.fch1 = (filhead.fch1 + filhead.foff / 2.0f) * 1e-06;
    //     filhead.nchans = FFTUSE;
    //     filhead.foff = -1.0 * filhead.foff / FFTUSE * 1e-06 ;
    
    //     filhead.fch1 = filhead.fch1 + filhead.foff / 2.0;
    
    //     if (DEBUG) {
    //         cout << "Some header info:\n"
    //                 << "Raw file: " << filhead.rawfile << endl
    //                 << "Source name: " << filhead.source << endl
    //                 << "Azimuth: " << filhead.az << endl
    //                 << "Zenith angle: " << filhead.za << endl
    //                 << "Declination: " << filhead.dec << endl
    //                 << "Right ascension: " << filhead.ra << endl
    //                 << "Top channel frequency: " << filhead.fch1 << endl
    //                 << "Channel bandwidth: " << filhead.foff << endl
    //                 << "Number of channels: " << filhead.nchans << endl
    //                 << "Sampling time: " << filhead.tsamp << endl
    //                 << "Bits per sample: " << filhead.nbits << endl;
    //     }
    
    //     // TODO: Make sure there are correct values for bandwidth and sampling time in the header after taking averaging into account
    
    //     ifstream filepola(inpola.c_str(), ifstream::in | ifstream::binary);
    //     ifstream filepolb(inpolb.c_str(), ifstream::in | ifstream::binary);
    //     ofstream filfile(outfil.c_str(), ofstream::out | ofstream::binary);
    
    //     if (!filepola || !filepolb) {
    //     if (!filepola) {
    //             cout << "Could not open file " << inpola << endl;
    //         }
    //         if (!filepolb) {
    //             cout << "Could not open file " << inpolb << endl;
    //         }
    //         exit(EXIT_FAILURE);
    //     }
    //     // TODO: Can save the filterbank header straight away, after the first header is read
    //     unsigned char vdifheadpola[32];
    //     unsigned char vdifheadpolb[32];
    //     filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //     filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    
    //     filepola.seekg(0, filepola.end);
    //     long long filelengtha = filepola.tellg();
    //     filepola.seekg(0, filepola.beg);
    
    //     filepolb.seekg(0, filepolb.end);
    //     long long filelengthb = filepolb.tellg();
    //     filepolb.seekg(0, filepolb.beg);
    
    //     unsigned int startframe;
    //     unsigned int startsecond;
    
    //     cout << filelengtha << " " << filelengthb << endl;
    
    //     startframe = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));	// frame number in this second
    //     startsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    
    //     if (DEBUG) {
    //         cout << "Starting time: " << startsecond << ":" << startframe << endl;
    //     }
    
    //     // NOTE: Need to read headers in
    //     unsigned int toread = NACCUMULATE * 8000;
    //     // NOTE: No more headers after unpacking
    //     unsigned int unpackedsize = NACCUMULATE * VDIFSIZE * UNPACKFACTOR;
    //     unsigned int fftedsize = unpackedsize / (2 * FFTUSE) * FFTOUT;
    //     unsigned int powersize = unpackedsize / (2 * FFTUSE) * FFTUSE / TIMEAVG;
    
    //     hipfftHandle fftplan;
    //     int fftsizes[1];
    //     fftsizes[0] = 2 * FFTUSE;
    //     int fftbatchsize = unpackedsize / fftsizes[0];
    //     cout << fftbatchsize << endl;
    //     cufftCheckError(hipfftPlanMany(&fftplan, 1, fftsizes, NULL, 1, FFTUSE, NULL, 1, FFTUSE, HIPFFT_R2C, fftbatchsize));
    
    //     unsigned char *tmppola = new unsigned char[toread];
    //     unsigned char *tmppolb = new unsigned char[toread];
    
    //     unsigned char *devpola;
    //     unsigned char *devpolb;
    //     unsigned char **datapol = new unsigned char*[NPOL];
    //     unsigned char **devpol;
    //     float **unpacked = new float*[NPOL];
    //     float **devunpacked;
    //     hipfftComplex **ffted = new hipfftComplex*[NPOL];
    //     hipfftComplex **devffted;
        
    //     unsigned char *devpower;
    //     unsigned char *tmppower = new unsigned char[powersize * filhead.nbits / 8];
    
    //     if (GPURUN) {
    //         cudaCheckError(hipMalloc((void**)&devpola, toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&devpolb, toread * sizeof(unsigned char)));
    
    //         cudaCheckError(hipMalloc((void**)&devpol, NPOL * sizeof(unsigned char*)));
    //         cudaCheckError(hipMalloc((void**)&datapol[0], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&datapol[1], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMemcpy(devpol, datapol, NPOL * sizeof(unsigned char*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devunpacked, NPOL * sizeof(float*)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[0], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[1], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMemcpy(devunpacked, unpacked, NPOL * sizeof(float*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devffted, NPOL * sizeof(hipfftComplex*)));
    //         cudaCheckError(hipMalloc((void**)&ffted[0], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMalloc((void**)&ffted[1], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMemcpy(devffted, ffted, NPOL * sizeof(hipfftComplex*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devpower, powersize * (filhead.nbits / 8)));
    //     }
    
    //     vector<std::pair<FrameInfo, FrameInfo>> vdifframes;
    
    //     FrameInfo tmpframea, tmpframeb;
    //     int refsecond;
    //     int frameno;
    //     int epoch;
    
    //     WriteFilterbankHeader(filfile, filhead);
       
    //     Timing runtimes;
    //     runtimes.readtime = 0.0f;
    //     runtimes.scaletime = 0.0f;
    //     runtimes.filtime = 0.0f;
    //     runtimes.savetime = 0.0f;
    //     runtimes.totaltime = 0.0f;
    //     runtimes.intertime = 0.0f;
    
    //     std::chrono::time_point<std::chrono::steady_clock> readstart, readend, scalestart, scaleend, filstart, filend, savestart, saveend, interstart, interend;
    
    //     float *tmpunpackeda = new float[unpackedsize];
    //     float *tmpunpackedb = new float[unpackedsize];
    //     hipfftComplex *tmpffta = new hipfftComplex[fftedsize];
    //     hipfftComplex *tmpfftb = new hipfftComplex[fftedsize];
    
    //     bool saved = false;
    
    //     //float *dmeans;
    //     //float *dstdevs;
    //     //cudaCheckError(hipMalloc((void**)&dmeans, FFTUSE * sizeof(float)));
    //     //cudaCheckError(hipMalloc((void**)&dstdevs, FFTUSE * sizeof(float)));
    
    //     thrust::device_vector<float> dmeans, dstdevs;
    //     dmeans.resize(FFTUSE);
    //     dstdevs.resize(FFTUSE);
    //     thrust::fill(dmeans.begin(), dmeans.end(), 0.0f);
    //     thrust::fill(dstdevs.begin(), dstdevs.end(), 0.0f);
    //     float *pdmeans = thrust::raw_pointer_cast(dmeans.data());
    //     float *pdstdevs = thrust::raw_pointer_cast(dstdevs.data());    
    
    //     cout << "Size of the device vectors: " << dmeans.size() << " " << dstdevs.size() << endl;
    
    //     scalestart = std::chrono::steady_clock::now();
    
    //     // NOTE: Use first 5 accumulates of data to obtain scaling factors
    //     if (scaling) {
    
    //         size_t divfactors = 5 * powersize / FFTUSE;
    //         thrust::device_vector<float> dfactors; 
    //         dfactors.resize(divfactors + 1);
    //         thrust::sequence(dfactors.begin(), dfactors.end());
    //         thrust::transform(dfactors.begin(), dfactors.end(), dfactors.begin(), FactorFunctor());
    //         float *pdfactors = thrust::raw_pointer_cast(dfactors.data());
    
    //         //float *dfactors;
    //         //size_t divfactors = 5 * powersize / FFTUSE;
    //         //cudaCheckError(hipMalloc((void**)&dfactors, divfactors * sizeof(float)));
    //         //int scalethreads = 1024;
    //         //int scaleblocks = (divfactors - 1) / scalethreads + 1;
    //         //cout << "Div factors blocks: " << scaleblocks << " and threads: " << scalethreads << endl;
    //         //InitDivFactors<<<scaleblocks, scalethreads>>>(dfactors, divfactors);
    //         //cudaCheckError(hipDeviceSynchronize());
    //         //cudaCheckError(hipGetLastError());
    //         size_t processed = 0;
    
    //         float *tmpdpower;
    //         cudaCheckError(hipMalloc((void**)&tmpdpower, powersize * sizeof(float)));
    
    //     while((filepola.tellg() < (5 * NACCUMULATE * 8032)) && (filepolb.tellg() < (5 * NACCUMULATE * 8032))) {
    //             for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //                 filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //                 filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //                 filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //                 filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             }
    
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    //             DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, tmpdpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             GetScalingFactorsKernel<<<1, FFTUSE, 0, 0>>>(tmpdpower, pdmeans, pdstdevs, pdfactors, processed);
    //             processed += (powersize / FFTUSE);
    //             cudaCheckError(hipDeviceSynchronize());
    //         }
    
    //         //float *hmeans = new float[FFTUSE];
    //         //float *hstdevs = new float[FFTUSE];
    
    //         //cudaCheckError(hipMemcpy(hmeans, dmeans, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    //         //cudaCheckError(hipMemcpy(hstdevs, dstdevs, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    
    //         thrust::host_vector<float> hmeans = dmeans;
    //         thrust::host_vector<float> hstdevs = dstdevs;
    
    //         std::ofstream statsfile("mean_stdev.dat");
    
    //         cout << "Size of host vector:" << hmeans.size() << endl;
     
    //         if (statsfile) {
    //             for (int ichan = 0; ichan < hmeans.size(); ++ichan) {
    //                 statsfile << hmeans[ichan] << " " << hstdevs[ichan] << endl;
    //             }
    //         } else {
    //             cerr << "Could not open the stats file" << endl;
    //         }
    
    //         statsfile.close();
    
    //         hipFree(tmpdpower);
             
    //     }
    
    //     scaleend = std::chrono::steady_clock::now();
    
    //     runtimes.scaletime = std::chrono::duration<float>(scaleend - scalestart).count();
    
    //     filepola.seekg(0, filepola.beg);
    //     filepolb.seekg(0, filepolb.beg);
    
    //     std::ofstream unpackedfilea ((outfil + ".unp0").c_str(), std::ios_base::binary);
    //     std::ofstream unpackedfileb ((outfil + ".unp1").c_str(), std::ios_base::binary);
    //     std::ofstream fftfilea ((outfil + ".fft0").c_str(), std::ios_base::binary);
    //     std::ofstream fftfileb ((outfil + ".fft1").c_str(), std::ios_base::binary);
    
    //     while((filepola.tellg() < (filelengtha - NACCUMULATE * 8000)) && (filepolb.tellg() < (filelengthb - NACCUMULATE * 8000))) {
    //         //cout << filepola.tellg() << endl;
    //         // NOTE: This implementation
    //         for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //         readstart = std::chrono::steady_clock::now();
    //             filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //             filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //             filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //             filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             readend = std::chrono::steady_clock::now();
    //             runtimes.readtime += std::chrono::duration<float>(readend - readstart).count();
    
    //             refsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));
    //             epoch = (unsigned int)(vdifheadpola[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframea.frameno = frameno;
    //             tmpframea.refsecond = refsecond;
    //             tmpframea.refepoch = epoch;
    
    //             refsecond = (unsigned int)(vdifheadpolb[0] | (vdifheadpolb[1] << 8) | (vdifheadpolb[2] << 16) | ((vdifheadpolb[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpolb[4] | (vdifheadpolb[5] << 8) | (vdifheadpolb[6] << 16));
    //             epoch = (unsigned int)(vdifheadpolb[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframeb.frameno = frameno;
    //             tmpframeb.refsecond = refsecond;
    //             tmpframeb.refepoch = epoch;
    
    //             vdifframes.push_back(std::make_pair(tmpframea, tmpframeb));
    
    //             // NOTE: Can use subtract startframe to put frame count at 0 and use that to save into the buffer
    
    //         }
     
    //        if (GPURUN) {
    //             filstart = std::chrono::steady_clock::now();
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    
    //             if (filhead.nbits == 8) {
    //                 DetectScaleKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<unsigned char*>(devpower), pdmeans, pdstdevs);
    //             } else if (filhead.nbits == 32) {
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             } else {
    //                 cerr << "Unsupported option! Will use float!" << endl;
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             }
    
    //             //PowerKernel<<<25, FFTUSE, 0, 0>>>(devffted, devpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             cudaCheckError(hipMemcpy(tmppower, devpower, powersize * filhead.nbits / 8, hipMemcpyDeviceToHost));
                
    //             if (!saved) {
    //                 std::ofstream unpackedfile("unpacked.dat");
    //         cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         for (int isamp = 0; isamp < 2 * 8000 * 4; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }
    //                 unpackedfile.close();
    //                 saved = true;
    //             }
    
    //             filend = std::chrono::steady_clock::now();
    //             runtimes.filtime += std::chrono::duration<float>(filend - filstart).count();
                
    //             savestart = std::chrono::steady_clock::now(); 
    //             filfile.write(reinterpret_cast<char*>(tmppower), powersize * filhead.nbits / 8);
    //             saveend = std::chrono::steady_clock::now();
    //             runtimes.savetime += std::chrono::duration<float>(saveend - savestart).count();
    
       
    
    //             if (saveinter) {
    
    //                 interstart = std::chrono::steady_clock::now();
    
    //                 cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < unpackedsize; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }*/
    
    //                 unpackedfilea.write(reinterpret_cast<char*>(tmpunpackeda), unpackedsize * sizeof(float));
    //                 unpackedfileb.write(reinterpret_cast<char*>(tmpunpackedb), unpackedsize * sizeof(float));
    
    //                 cudaCheckError(hipMemcpy(tmpffta, ffted[0], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 cudaCheckError(hipMemcpy(tmpfftb, ffted[1], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < fftedsize; ++isamp) {
    //                     fftfile << tmpffta[isamp].x << " " << tmpffta[isamp].y << " " << tmpfftb[isamp].x << " " << tmpfftb[isamp].y << endl;
    //                 }*/
    
    //                 fftfilea.write(reinterpret_cast<char*>(tmpffta), fftedsize * sizeof(hipfftComplex));
    //                 fftfileb.write(reinterpret_cast<char*>(tmpfftb), fftedsize * sizeof(hipfftComplex));
    
    //                 interend = std::chrono::steady_clock::now();
    //                 runtimes.intertime += std::chrono::duration<float>(interend - interstart).count();           
    
    //             }
    
    //         }
    //         cout << "Completed " << std::fixed << std::setprecision(2) << (float)filepola.tellg() / (float)(filelengtha - 1.0) * 100.0f << "%\r";
    //         cout.flush();
    //     }
    
    //     cout << endl;
    //     filfile.close();
    //     unpackedfilea.close();
    //     unpackedfileb.close();
    //     fftfilea.close();
    //     fftfileb.close();
    
    //     runtimes.totaltime = runtimes.readtime + runtimes.scaletime + runtimes.filtime + runtimes.savetime + runtimes.intertime;
    
    //     cout << "Total execution time: " << runtimes.totaltime << "s\n";
    //     cout << "\tScaling factors: " << runtimes.scaletime << "s\n";
    //     cout << "\tFile read: " << runtimes.readtime << "s\n";
    //     cout << "\tFilterbanking: " << runtimes.filtime << "s\n";
    //     cout << "\tFile write: " << runtimes.savetime << "s\n";
    //     if (saveinter) {
    //         cout << "\tIntermediate write: " << runtimes.intertime << "s\n";
    //     }
    
    //     if (DEBUG) {
    //         std::ofstream outframes("dataframes.dat");
    
    //         outframes << "Ref Epoch A\tRef second A\tRef frame A\tRef Epoch B\tRef second B\tRef frame b\n";
    //         for (auto iframe = vdifframes.begin(); iframe != vdifframes.end(); ++iframe) {
    //             outframes << iframe->first.refepoch << "\t" << iframe->first.refsecond << "\t" << iframe->first.frameno << "\t"
    //             << iframe->second.refepoch << "\t" << iframe->second.refsecond << "\t" << iframe->second.frameno << endl;
    //         }
    
    //         outframes.close();
    //     }


    // }

    return 0;
}
