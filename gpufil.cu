#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <string>
#include <utility>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include "constants.hpp"
#include "dada.hpp"
#include "errors.hpp"
#include "filterbank.hpp"
#include "kernels.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::ifstream;
using std::ofstream;
using std::string;
using std::vector;

struct FrameInfo {
    unsigned int frameno;
    unsigned int refsecond;
    unsigned int refepoch;
};

struct Timing {
    float readtime;
    float scaletime;
    float filtime;
    float savetime;
    float totaltime;
    float intertime;
};
 /*
struct FactorFunctor {
    __host__ __device__ float operator()(float val) {
        return val != 0 ? 1.0f/val : val;
    }
};

__constant__ unsigned char kMask[] = {0x03, 0x0C, 0x30, 0xC0};

__global__ void UnpackKernel(unsigned char **in, float **out, size_t samples) {

    // NOTE: Each thread in the block processes 625 incoming bytes
    int idx = blockIdx.x * blockDim.x * PERBLOCK + threadIdx.x;
    int tmod = threadIdx.x % 4;

    // NOTE: Each thread can store one value
    __shared__ unsigned char incoming[1024];

    int outidx = blockIdx.x * blockDim.x * PERBLOCK * 4;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {
        if (idx < samples) {
            for (int ipol = 0; ipol < 2; ++ipol) {
                incoming[threadIdx.x] = in[ipol][idx];
                __syncthreads();
                int outidx2 = outidx + threadIdx.x;
		for (int ichunk = 0; ichunk < 4; ++ichunk) {
                    int inidx = threadIdx.x / 4 + ichunk * 256;
                    unsigned char inval = incoming[inidx];
                    out[ipol][outidx2] = static_cast<float>(static_cast<short>(((inval & kMask[tmod]) >> (2 * tmod))));
                    outidx2 += 1024;
                }
            }
        }
        idx += blockDim.x;
        outidx += blockDim.x * 4;
    }
}

// NOTE: Does not do any frequency averaging
// NOTE: Outputs only the total intensity and no other Stokes parameters
// NOTE: PERBLOCK is the number of output samples per block
__global__ void DetectKernel(hipfftComplex** __restrict__ in, float* __restrict__ out) {
    int outidx = blockIdx.x * PERBLOCK * FFTUSE + FFTUSE - threadIdx.x - 1;
    int inidx = blockIdx.x * PERBLOCK * TIMEAVG * FFTOUT + threadIdx.x + 1;

    float outvalue = 0.0f;
    hipfftComplex polval;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {

        // NOTE: Read the data from the incoming array
        for (int ipol = 0; ipol < 2; ++ipol) {
            for (int iavg = 0; iavg < TIMEAVG; ++iavg) {
                polval = in[ipol][inidx + iavg * FFTOUT];
                outvalue += polval.x * polval.x + polval.y * polval.y;
            }

        }
        outvalue *= TIMESCALE;
        out[outidx] = outvalue;
        inidx += FFTOUT * TIMEAVG;
        outidx += FFTUSE;
        outvalue = 0.0;
    }
}

__global__ void DetectScaleKernel(hipfftComplex** __restrict__ in, unsigned char* __restrict__ out, float* __restrict__ means, float* __restrict__ stdevs) {
    int outidx = blockIdx.x * PERBLOCK * FFTUSE + FFTUSE - threadIdx.x - 1;
    int inidx = blockIdx.x * PERBLOCK * TIMEAVG * FFTOUT + threadIdx.x + 1;

    float outvalue = 0.0f;
    hipfftComplex polval;

    int scaled = 0;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {

        // NOTE: Read the data from the incoming array
        for (int ipol = 0; ipol < 2; ++ipol) {
            for (int iavg = 0; iavg < TIMEAVG; ++iavg) {
                polval = in[ipol][inidx + iavg * FFTOUT];
                outvalue += polval.x * polval.x + polval.y * polval.y;
            }

        }
        outvalue *= TIMESCALE;
        scaled = __float2int_ru((outvalue - means[FFTUSE - threadIdx.x - 1]) / stdevs[FFTUSE - threadIdx.x - 1] * 32.0f + 128.0f);
        if (scaled > 255) {
            scaled = 255;
        } else if (scaled < 0) {
            scaled = 0;
        }
        out[outidx] = (unsigned char)scaled;
        inidx += FFTOUT * TIMEAVG;
        outidx += FFTUSE;
        outvalue = 0.0;
    }
}

__global__ void InitDivFactors(float *factors, size_t togenerate) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // NOTE: I don't want to be dividing by 0
    // NOTE: idx of 0 will not be used anyway
    if (idx < togenerate) {
        if (idx != 0) {
            factors[idx] = 1.0f / idx;
        } else {
            factors[idx] = idx;
        }
    }
}

__global__ void GetScalingFactorsKernel(float* __restrict__ indata, float *base, float *stdev, float *factors, int processed) {

    // NOTE: Filterbank file format coming in
    //float mean = indata[threadIdx.x];
    float mean = 0.0f;
    // NOTE: Depending whether I save STD or VAR at the end of every run
    // float estd = stdev[threadIdx.x];
    float estd = stdev[threadIdx.x] * stdev[threadIdx.x] * (processed - 1.0f);
    float oldmean = base[threadIdx.x];

    //float estd = 0.0f;
    //float oldmean = 0.0;

    float val = 0.0f;
    float diff = 0.0;
    // NOTE: There are 15625 output time samples per NACCUMULATE frames
    for (int isamp = 0; isamp < 15625; ++isamp) {
        val = indata[isamp * FFTUSE + threadIdx.x];
        diff = val - oldmean;
        mean = oldmean + diff * factors[processed + isamp + 1];
        estd += diff * (val - mean);
        oldmean = mean;
    }
    base[threadIdx.x] = mean;
    stdev[threadIdx.x] = sqrtf(estd / (float)(processed + 15625 - 1.0f));
    // stdev[threadIdx.x] = estd;
}
*/



int main(int argc, char *argv[]) {

    string inpola = "";
    string inpolb = "";
    string outfil = "";
    string config = "";
    string dadastr = "";
    double readsec; 
    bool scaling = false;
    bool saveinter = false;

    std::vector<std::string> dadastrings; 

    if ((argc < 5) || (argv[1] == "-h") || (argv[1] == "--help")) {
        cout << "Incorrect number of arguments!" << endl;
        cout << "Command line options:" << endl
                << "-a <filename> - input file for polarisation a" << endl
                << "-b <filename> - input file for polarisation b" << endl
                << "-d <filename> - input DADA file" << endl
                << "-o <filename> - output filterbank file" << endl
                << "-c <filename> - input configuration file" << endl
                << "-r <number> - number of seconds to process - CURRENTLY NOT WORKING" << endl
                << "-s - enable scaling from 32 bits to 8 bits" << endl
                << "-i - enable saving the intermediate data products" << endl
                << "-h, --help - display this message" << endl;
        exit(EXIT_SUCCESS);
    }

    for (int iarg = 0; iarg < argc; ++iarg) {
        if (std::string(argv[iarg]) == "-a") {
            iarg++;
            inpola = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-b") {
            iarg++;
            inpolb = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-d") {
            for (int ifile = 0; ifile < 4; ++ifile) {
                iarg++;
                dadastr = std::string(argv[iarg]);
                dadastrings.push_back(dadastr);
            }
        } else if (std::string(argv[iarg]) == "-o") {
            iarg++;
            outfil = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-c") {
            iarg++;
            config = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-s") {
            cout << "Will scale the data to 8 bits" << endl;
            scaling = true;
        } else if (std::string(argv[iarg]) == "-i") {
            cout << "Will save the intermediate products" << endl;
            saveinter = true;
        } else if (std::string(argv[iarg]) == "-r") {
            iarg++;
            readsec = std::stod(argv[iarg]);
        }
    }

    if (!inpola.empty() && !dadastr.empty()) {
        cerr << "It's one or the other: DADA or VDIF, not both!" << endl;
        return 1;
    }

    if (!dadastrings.empty()) {

        std::cout << "Input files: ";
            for (auto &dadastring: dadastrings) {
                std::cout << dadastring << " ";
            }
        std::cout << std::endl;
        
        long long filesize = 0;

        for (auto &dadastring: dadastrings) {
            std::ifstream indada(dadastring.c_str(), std::ios_base::binary);

            if (indada) {

                indada.seekg(0, indada.end);
                if (!filesize) {
                    filesize = indada.tellg() - 4096L;
                }
                
                if (filesize != indada.tellg() - 4096L) {
                    std::cerr << "Files do not have the same size!" << std::endl;
                    exit(EXIT_FAILURE);
                }
                indada.close();

            } else {
                std::cerr << "Could not open file: " << dadastring << std::endl;
                exit(EXIT_FAILURE);
            }

        }

        /* std::ifstream indada(dadastr.c_str(), std::ios_base::binary);        
        long long filesize = 0;
        indada.seekg(0, indada.end);
        filesize = indada.tellg() - 4096L;
        indada.seekg(0, indada.beg);
        */ 
        // NOTE: 4 bytes per full time sample: 1 byte sampling, 2 polarisations, complex number
        size_t totalsamples = filesize / 4;
        if (filesize != totalsamples * 4) {
            std::cerr << "A non-integer number of time samples was read - something went very wrong!" << std::endl;
            return 1;
        }

        std::cout << "File size: " << filesize / 1024.0f / 1024.0f << "MiB with " << totalsamples << " time samples" << std::endl;
        // NOTE: That simply ensures that we only process the integer number of final filterbank channels
        totalsamples = (int)((float)totalsamples / (OUTCHANS * TIMEAVG)) * OUTCHANS * TIMEAVG;
        std::cout << "Will use first" << totalsamples << " samples" << std::endl;

        size_t freemem = 0;
        size_t totalmem = 0;
        cudaCheckError(hipMemGetInfo(&freemem, &totalmem));
        // NOTE: Let's liffh just 25% of what's free, because cuFFT happens...
        freemem = freemem * 0.25;
        std::cout << "Total memory: " << totalmem / 1024.0f / 1024.0f << "MiB, with " << freemem / 1024.0f / 1024.0f << "MiB free" << std::endl;
        
        // original file + original file cast to hipfftComplex for FFT + output filterbank file saved as 32 bit float, all times the number of input files
        size_t needmem = (4 * totalsamples + 4 * totalsamples * 4 + totalsamples / OUTCHANS / TIMEAVG * OUTCHANS * 4) * dadastrings.size();
        std::cout << "Need " << needmem / 1024.0f / 1024.0f << "MiB on the device" << std::endl;
        
        int nblocks = 0;
        size_t sampperblock = 0;
        size_t remsamp = 0;
        
        if (needmem < freemem) {
            std::cout << "Can store everything in global memory at once..." << std::endl;
            nblocks = 1;
            sampperblock = totalsamples;
        } else {
            std::cout << "We need to divide the job..." << std::endl;

            sampperblock = (int)((float)freemem / (dadastrings.size() * (float)(OUTCHANS * TIMEAVG) * (4.0f + 16.0f + 4.0f / (float)TIMEAVG))) * OUTCHANS * TIMEAVG;
            nblocks = (int)(totalsamples / sampperblock);
            remsamp = totalsamples - nblocks * sampperblock;

            std::cout << "Will process the data in " << nblocks << " blocks, with "
                        << sampperblock << " samples per block "
                        << "(" << dadastrings.size() << " files per block)";
            if (remsamp) {
                std::cout << " and an extra block with " << remsamp << " samples at the end";
            }
            std::cout << std::endl;
        }

        /**** ####
        // STAGE: MEMORY AND FFT
        #### ****/
        // NOTE: Factor of 4 to account for 2 polarisations and complex components for every time sample
        size_t blockread = sampperblock * 4 * dadastrings.size();
        size_t perfileread = sampperblock * 4;
        size_t remread = remsamp * 4 * dadastrings.size();
        size_t perfilerem = remsamp * 4;
        
        // NOTE: This is a very annoying stage where hipfftPlanMany uses ridiculous amount of temporary buffer and runs out of memory most of the time
        hipfftHandle fftplan;
        int fftsizes[1];
        fftsizes[0] = OUTCHANS;
        // NOTE: Factor of 2 to account for 2 polarisations
        int fftbatchsize = sampperblock * 2 / fftsizes[0] * dadastrings.size();
        cufftCheckError(hipfftPlanMany(&fftplan, 1, fftsizes, NULL, 1, OUTCHANS, NULL, 1, OUTCHANS, HIPFFT_C2C, fftbatchsize));
 
        unsigned char *hostvoltage = new unsigned char[blockread];
        unsigned char *devicevoltage = new unsigned char[blockread];
        cudaCheckError(hipMalloc((void**)&devicevoltage, blockread * sizeof(unsigned char)));

        hipfftComplex *devicefft;
        cudaCheckError(hipMalloc((void**)&devicefft, sampperblock * 2 * sizeof(hipfftComplex)));

        size_t powersize = sampperblock / OUTCHANS * OUTCHANS / TIMEAVG * dadastrings.size();
        float *hostpower = new float[powersize];
        float *devicepower;
        cudaCheckError(hipMalloc((void**)&devicepower, powersize * sizeof(float)))

        float *hostband = new float[OUTCHANS * dadastrings.size()];
        float *deviceband;
        cudaCheckError(hipMalloc((void**)&deviceband, OUTCHANS * dadastrings.size() * sizeof(float)));
        

        size_t fullfillsize = powersize * dadastrings.size() + remsamp / OUTCHANS / TIMEAVG * OUTCHANS;
        float *fullfil = new float[fullfillsize];

        std::vector<FilHead> filheaders;
        std::vector<std::ifstream> dadastreams;

        for (int ifile = 0; ifile < dadastrings.size(); ++ifile) {

            dadastreams.push_back(std::ifstream());

            dadastreams.back().open(dadastrings.at(ifile).c_str(), std::ios_base::binary);

            // std::ifstream indada(dadastrings.at(ifile).c_str(), std::ios_base::binary);
            
            FilHead filhead = {};
            ReadDadaHeader(dadastreams.back(), filhead);

            if (!scaling) {
                filhead.nbits = 32;
            }
            filhead.nchans = OUTCHANS;
            filhead.tsamp = filhead.tsamp * OUTCHANS * TIMEAVG;
        
            filheaders.push_back(filhead);
        
            PrintFilterbankHeader(filheaders.at(ifile));
            
            // NOTE: Just in case I did something wrong
            dadastreams.back().seekg(4096, dadastreams.back().beg);

        }

        std::ofstream filfile(outfil.c_str(), std::ios_base::binary);
        //WriteFilterbankHeader(filfile, filhead);

        for (int iblock = 0; iblock < nblocks; iblock++) {

            std::cout << "Processing block " << iblock << "..." << std::endl;

            for (int ifile = 0; ifile < dadastrings.size(); ++ifile) {
                std::cout << "Reading file " << dadastrings.at(ifile) << "..." << std::endl;
               
                dadastreams.at(ifile).read(reinterpret_cast<char*>(hostvoltage + ifile * perfileread), perfileread * sizeof(unsigned char));
            }

            cudaCheckError(hipMemcpy(devicevoltage, hostvoltage, blockread * sizeof(unsigned char), hipMemcpyHostToDevice));

            dim3 block (OUTCHANS, 1, 1);
            dim3 grid (64, 1, 1);

            UnpackDadaKernel<<<grid, block, 0, 0>>>(sampperblock * dadastrings.size(), reinterpret_cast<uchar4*>(devicevoltage), devicefft);
            cudaCheckError(hipGetLastError());

            cufftCheckError(hipfftExecC2C(fftplan, devicefft, devicefft, HIPFFT_FORWARD));

            DetectDadaKernel<<<grid, block, 0, 0>>>(sampperblock / OUTCHANS, devicefft, devicepower);
            cudaCheckError(hipGetLastError());

            BandpassKernel<<<1, OUTCHANS, 0, 0>>>(sampperblock / OUTCHANS / TIMEAVG, devicepower, deviceband);
            cudaCheckError(hipGetLastError());

            //cudaCheckError(hipMemcpy(hostpower, devicepower, powersize * sizeof(float), hipMemcpyDeviceToHost));

            //filfile.write(reinterpret_cast<char*>(hostpower), powersize * sizeof(float));

            cudaCheckError(hipMemcpy(fullfil + powersize * dadastrings.size() * iblock, devicepower,
                                        powersize * dadastrings.size() * sizeof(float), hipMemcpyDeviceToHost));
        } 
        
        cufftCheckError(hipfftDestroy(fftplan));

        if (remsamp) {

            std::cout << "Processing the remainder block..." << std::endl;

            for (int ifile = 0; ifile < dadastrings.size(); ++ifile) {
                std::cout << "Reading file " << dadastrings.at(ifile) << "..." << std::endl;
               
                dadastreams.at(ifile).read(reinterpret_cast<char*>(hostvoltage + ifile * perfilerem), perfilerem * sizeof(unsigned char));
            }

            cudaCheckError(hipMemcpy(devicevoltage, hostvoltage, remread * sizeof(unsigned char), hipMemcpyHostToDevice));

            dim3 block(OUTCHANS, 1, 1);
            dim3 grid (64, 1, 1);

            UnpackDadaKernel<<<grid, block, 0, 0>>>(remsamp, reinterpret_cast<uchar4*>(devicevoltage), devicefft);
            cudaCheckError(hipGetLastError());

            hipfftHandle fftplanrem;
            int fftrembatchsize = remsamp * 2 / fftsizes[0];
            cufftCheckError(hipfftPlanMany(&fftplanrem, 1, fftsizes, NULL, 1, OUTCHANS, NULL, 1, OUTCHANS, HIPFFT_C2C, fftrembatchsize));

            cufftCheckError(hipfftExecC2C(fftplanrem, devicefft, devicefft, HIPFFT_FORWARD));

            DetectDadaKernel<<<grid, block, 0, 0>>>(remsamp / OUTCHANS, devicefft, devicepower);
            cudaCheckError(hipGetLastError());

            BandpassKernel<<<1, OUTCHANS, 0, 0>>>(remsamp / OUTCHANS / TIMEAVG, devicepower, deviceband);
            cudaCheckError(hipGetLastError());

            //cudaCheckError(hipMemcpy(hostpower, devicepower, remsamp / OUTCHANS / TIMEAVG * OUTCHANS * sizeof(float), hipMemcpyDeviceToHost));

            //filfile.write(reinterpret_cast<char*>(hostpower), remsamp / OUTCHANS / TIMEAVG * OUTCHANS * sizeof(float));

            cufftCheckError(hipfftDestroy(fftplanrem));

            cudaCheckError(hipMemcpy(fullfil + nblocks * powersize * dadastrings.size(), devicepower,
                                        remsamp / OUTCHANS / TIMEAVG * OUTCHANS * sizeof(float) * dadastrings.size(), hipMemcpyDeviceToHost));
        }

        cudaCheckError(hipMemcpy(hostband, deviceband, OUTCHANS * sizeof(float), hipMemcpyDeviceToHost));



        std::ofstream bandout("bandpass.dat");

        if (bandout) {
            for (int ichan = 0; ichan < OUTCHANS; ++ichan) {
                bandout << hostband[ichan] << std::endl;
            }
        }

        bandout.close();
        filfile.close();
        
        for (auto &dadastream: dadastreams) {
            dadastream.close();
        }

        hipFree(deviceband);
        hipFree(devicepower);
        hipFree(devicefft);
        hipFree(devicevoltage);

        delete [] fullfil;
        delete [] hostband;
        delete [] hostpower;
        delete [] hostvoltage;

    } 

    // NOTE: This code is in a bit of a bad state - need to clean it up
    // else if (!inpola.empty() & !inpolb.empty()) {

    //     cout << "Input files: " << inpola << " " << inpolb << endl;

    //     FilHead filhead;
    //     ReadFilterbankHeader(config, filhead);



        
    //     if (scaling) {
    //         filhead.nbits = 8;
    //     }
    
    //     // TODO: This will be wrong for R2C FFT
    //     filhead.tsamp = 1.0 / (2.0 * filhead.foff) * 2 * FFTUSE * TIMEAVG;
    //     // TODO: Make sure it is the middle of the top frequency channel
    //     filhead.fch1 = (filhead.fch1 + filhead.foff / 2.0f) * 1e-06;
    //     filhead.nchans = FFTUSE;
    //     filhead.foff = -1.0 * filhead.foff / FFTUSE * 1e-06 ;
    
    //     filhead.fch1 = filhead.fch1 + filhead.foff / 2.0;
    
    //     if (DEBUG) {
    //         cout << "Some header info:\n"
    //                 << "Raw file: " << filhead.rawfile << endl
    //                 << "Source name: " << filhead.source << endl
    //                 << "Azimuth: " << filhead.az << endl
    //                 << "Zenith angle: " << filhead.za << endl
    //                 << "Declination: " << filhead.dec << endl
    //                 << "Right ascension: " << filhead.ra << endl
    //                 << "Top channel frequency: " << filhead.fch1 << endl
    //                 << "Channel bandwidth: " << filhead.foff << endl
    //                 << "Number of channels: " << filhead.nchans << endl
    //                 << "Sampling time: " << filhead.tsamp << endl
    //                 << "Bits per sample: " << filhead.nbits << endl;
    //     }
    
    //     // TODO: Make sure there are correct values for bandwidth and sampling time in the header after taking averaging into account
    
    //     ifstream filepola(inpola.c_str(), ifstream::in | ifstream::binary);
    //     ifstream filepolb(inpolb.c_str(), ifstream::in | ifstream::binary);
    //     ofstream filfile(outfil.c_str(), ofstream::out | ofstream::binary);
    
    //     if (!filepola || !filepolb) {
    //     if (!filepola) {
    //             cout << "Could not open file " << inpola << endl;
    //         }
    //         if (!filepolb) {
    //             cout << "Could not open file " << inpolb << endl;
    //         }
    //         exit(EXIT_FAILURE);
    //     }
    //     // TODO: Can save the filterbank header straight away, after the first header is read
    //     unsigned char vdifheadpola[32];
    //     unsigned char vdifheadpolb[32];
    //     filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //     filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    
    //     filepola.seekg(0, filepola.end);
    //     long long filelengtha = filepola.tellg();
    //     filepola.seekg(0, filepola.beg);
    
    //     filepolb.seekg(0, filepolb.end);
    //     long long filelengthb = filepolb.tellg();
    //     filepolb.seekg(0, filepolb.beg);
    
    //     unsigned int startframe;
    //     unsigned int startsecond;
    
    //     cout << filelengtha << " " << filelengthb << endl;
    
    //     startframe = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));	// frame number in this second
    //     startsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    
    //     if (DEBUG) {
    //         cout << "Starting time: " << startsecond << ":" << startframe << endl;
    //     }
    
    //     // NOTE: Need to read headers in
    //     unsigned int toread = NACCUMULATE * 8000;
    //     // NOTE: No more headers after unpacking
    //     unsigned int unpackedsize = NACCUMULATE * VDIFSIZE * UNPACKFACTOR;
    //     unsigned int fftedsize = unpackedsize / (2 * FFTUSE) * FFTOUT;
    //     unsigned int powersize = unpackedsize / (2 * FFTUSE) * FFTUSE / TIMEAVG;
    
    //     hipfftHandle fftplan;
    //     int fftsizes[1];
    //     fftsizes[0] = 2 * FFTUSE;
    //     int fftbatchsize = unpackedsize / fftsizes[0];
    //     cout << fftbatchsize << endl;
    //     cufftCheckError(hipfftPlanMany(&fftplan, 1, fftsizes, NULL, 1, FFTUSE, NULL, 1, FFTUSE, HIPFFT_R2C, fftbatchsize));
    
    //     unsigned char *tmppola = new unsigned char[toread];
    //     unsigned char *tmppolb = new unsigned char[toread];
    
    //     unsigned char *devpola;
    //     unsigned char *devpolb;
    //     unsigned char **datapol = new unsigned char*[NPOL];
    //     unsigned char **devpol;
    //     float **unpacked = new float*[NPOL];
    //     float **devunpacked;
    //     hipfftComplex **ffted = new hipfftComplex*[NPOL];
    //     hipfftComplex **devffted;
        
    //     unsigned char *devpower;
    //     unsigned char *tmppower = new unsigned char[powersize * filhead.nbits / 8];
    
    //     if (GPURUN) {
    //         cudaCheckError(hipMalloc((void**)&devpola, toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&devpolb, toread * sizeof(unsigned char)));
    
    //         cudaCheckError(hipMalloc((void**)&devpol, NPOL * sizeof(unsigned char*)));
    //         cudaCheckError(hipMalloc((void**)&datapol[0], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&datapol[1], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMemcpy(devpol, datapol, NPOL * sizeof(unsigned char*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devunpacked, NPOL * sizeof(float*)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[0], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[1], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMemcpy(devunpacked, unpacked, NPOL * sizeof(float*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devffted, NPOL * sizeof(hipfftComplex*)));
    //         cudaCheckError(hipMalloc((void**)&ffted[0], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMalloc((void**)&ffted[1], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMemcpy(devffted, ffted, NPOL * sizeof(hipfftComplex*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devpower, powersize * (filhead.nbits / 8)));
    //     }
    
    //     vector<std::pair<FrameInfo, FrameInfo>> vdifframes;
    
    //     FrameInfo tmpframea, tmpframeb;
    //     int refsecond;
    //     int frameno;
    //     int epoch;
    
    //     WriteFilterbankHeader(filfile, filhead);
       
    //     Timing runtimes;
    //     runtimes.readtime = 0.0f;
    //     runtimes.scaletime = 0.0f;
    //     runtimes.filtime = 0.0f;
    //     runtimes.savetime = 0.0f;
    //     runtimes.totaltime = 0.0f;
    //     runtimes.intertime = 0.0f;
    
    //     std::chrono::time_point<std::chrono::steady_clock> readstart, readend, scalestart, scaleend, filstart, filend, savestart, saveend, interstart, interend;
    
    //     float *tmpunpackeda = new float[unpackedsize];
    //     float *tmpunpackedb = new float[unpackedsize];
    //     hipfftComplex *tmpffta = new hipfftComplex[fftedsize];
    //     hipfftComplex *tmpfftb = new hipfftComplex[fftedsize];
    
    //     bool saved = false;
    
    //     //float *dmeans;
    //     //float *dstdevs;
    //     //cudaCheckError(hipMalloc((void**)&dmeans, FFTUSE * sizeof(float)));
    //     //cudaCheckError(hipMalloc((void**)&dstdevs, FFTUSE * sizeof(float)));
    
    //     thrust::device_vector<float> dmeans, dstdevs;
    //     dmeans.resize(FFTUSE);
    //     dstdevs.resize(FFTUSE);
    //     thrust::fill(dmeans.begin(), dmeans.end(), 0.0f);
    //     thrust::fill(dstdevs.begin(), dstdevs.end(), 0.0f);
    //     float *pdmeans = thrust::raw_pointer_cast(dmeans.data());
    //     float *pdstdevs = thrust::raw_pointer_cast(dstdevs.data());    
    
    //     cout << "Size of the device vectors: " << dmeans.size() << " " << dstdevs.size() << endl;
    
    //     scalestart = std::chrono::steady_clock::now();
    
    //     // NOTE: Use first 5 accumulates of data to obtain scaling factors
    //     if (scaling) {
    
    //         size_t divfactors = 5 * powersize / FFTUSE;
    //         thrust::device_vector<float> dfactors; 
    //         dfactors.resize(divfactors + 1);
    //         thrust::sequence(dfactors.begin(), dfactors.end());
    //         thrust::transform(dfactors.begin(), dfactors.end(), dfactors.begin(), FactorFunctor());
    //         float *pdfactors = thrust::raw_pointer_cast(dfactors.data());
    
    //         //float *dfactors;
    //         //size_t divfactors = 5 * powersize / FFTUSE;
    //         //cudaCheckError(hipMalloc((void**)&dfactors, divfactors * sizeof(float)));
    //         //int scalethreads = 1024;
    //         //int scaleblocks = (divfactors - 1) / scalethreads + 1;
    //         //cout << "Div factors blocks: " << scaleblocks << " and threads: " << scalethreads << endl;
    //         //InitDivFactors<<<scaleblocks, scalethreads>>>(dfactors, divfactors);
    //         //cudaCheckError(hipDeviceSynchronize());
    //         //cudaCheckError(hipGetLastError());
    //         size_t processed = 0;
    
    //         float *tmpdpower;
    //         cudaCheckError(hipMalloc((void**)&tmpdpower, powersize * sizeof(float)));
    
    //     while((filepola.tellg() < (5 * NACCUMULATE * 8032)) && (filepolb.tellg() < (5 * NACCUMULATE * 8032))) {
    //             for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //                 filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //                 filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //                 filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //                 filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             }
    
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    //             DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, tmpdpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             GetScalingFactorsKernel<<<1, FFTUSE, 0, 0>>>(tmpdpower, pdmeans, pdstdevs, pdfactors, processed);
    //             processed += (powersize / FFTUSE);
    //             cudaCheckError(hipDeviceSynchronize());
    //         }
    
    //         //float *hmeans = new float[FFTUSE];
    //         //float *hstdevs = new float[FFTUSE];
    
    //         //cudaCheckError(hipMemcpy(hmeans, dmeans, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    //         //cudaCheckError(hipMemcpy(hstdevs, dstdevs, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    
    //         thrust::host_vector<float> hmeans = dmeans;
    //         thrust::host_vector<float> hstdevs = dstdevs;
    
    //         std::ofstream statsfile("mean_stdev.dat");
    
    //         cout << "Size of host vector:" << hmeans.size() << endl;
     
    //         if (statsfile) {
    //             for (int ichan = 0; ichan < hmeans.size(); ++ichan) {
    //                 statsfile << hmeans[ichan] << " " << hstdevs[ichan] << endl;
    //             }
    //         } else {
    //             cerr << "Could not open the stats file" << endl;
    //         }
    
    //         statsfile.close();
    
    //         hipFree(tmpdpower);
             
    //     }
    
    //     scaleend = std::chrono::steady_clock::now();
    
    //     runtimes.scaletime = std::chrono::duration<float>(scaleend - scalestart).count();
    
    //     filepola.seekg(0, filepola.beg);
    //     filepolb.seekg(0, filepolb.beg);
    
    //     std::ofstream unpackedfilea ((outfil + ".unp0").c_str(), std::ios_base::binary);
    //     std::ofstream unpackedfileb ((outfil + ".unp1").c_str(), std::ios_base::binary);
    //     std::ofstream fftfilea ((outfil + ".fft0").c_str(), std::ios_base::binary);
    //     std::ofstream fftfileb ((outfil + ".fft1").c_str(), std::ios_base::binary);
    
    //     while((filepola.tellg() < (filelengtha - NACCUMULATE * 8000)) && (filepolb.tellg() < (filelengthb - NACCUMULATE * 8000))) {
    //         //cout << filepola.tellg() << endl;
    //         // NOTE: This implementation
    //         for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //         readstart = std::chrono::steady_clock::now();
    //             filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //             filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //             filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //             filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             readend = std::chrono::steady_clock::now();
    //             runtimes.readtime += std::chrono::duration<float>(readend - readstart).count();
    
    //             refsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));
    //             epoch = (unsigned int)(vdifheadpola[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframea.frameno = frameno;
    //             tmpframea.refsecond = refsecond;
    //             tmpframea.refepoch = epoch;
    
    //             refsecond = (unsigned int)(vdifheadpolb[0] | (vdifheadpolb[1] << 8) | (vdifheadpolb[2] << 16) | ((vdifheadpolb[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpolb[4] | (vdifheadpolb[5] << 8) | (vdifheadpolb[6] << 16));
    //             epoch = (unsigned int)(vdifheadpolb[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframeb.frameno = frameno;
    //             tmpframeb.refsecond = refsecond;
    //             tmpframeb.refepoch = epoch;
    
    //             vdifframes.push_back(std::make_pair(tmpframea, tmpframeb));
    
    //             // NOTE: Can use subtract startframe to put frame count at 0 and use that to save into the buffer
    
    //         }
     
    //        if (GPURUN) {
    //             filstart = std::chrono::steady_clock::now();
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    
    //             if (filhead.nbits == 8) {
    //                 DetectScaleKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<unsigned char*>(devpower), pdmeans, pdstdevs);
    //             } else if (filhead.nbits == 32) {
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             } else {
    //                 cerr << "Unsupported option! Will use float!" << endl;
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             }
    
    //             //PowerKernel<<<25, FFTUSE, 0, 0>>>(devffted, devpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             cudaCheckError(hipMemcpy(tmppower, devpower, powersize * filhead.nbits / 8, hipMemcpyDeviceToHost));
                
    //             if (!saved) {
    //                 std::ofstream unpackedfile("unpacked.dat");
    //         cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         for (int isamp = 0; isamp < 2 * 8000 * 4; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }
    //                 unpackedfile.close();
    //                 saved = true;
    //             }
    
    //             filend = std::chrono::steady_clock::now();
    //             runtimes.filtime += std::chrono::duration<float>(filend - filstart).count();
                
    //             savestart = std::chrono::steady_clock::now(); 
    //             filfile.write(reinterpret_cast<char*>(tmppower), powersize * filhead.nbits / 8);
    //             saveend = std::chrono::steady_clock::now();
    //             runtimes.savetime += std::chrono::duration<float>(saveend - savestart).count();
    
       
    
    //             if (saveinter) {
    
    //                 interstart = std::chrono::steady_clock::now();
    
    //                 cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < unpackedsize; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }*/
    
    //                 unpackedfilea.write(reinterpret_cast<char*>(tmpunpackeda), unpackedsize * sizeof(float));
    //                 unpackedfileb.write(reinterpret_cast<char*>(tmpunpackedb), unpackedsize * sizeof(float));
    
    //                 cudaCheckError(hipMemcpy(tmpffta, ffted[0], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 cudaCheckError(hipMemcpy(tmpfftb, ffted[1], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < fftedsize; ++isamp) {
    //                     fftfile << tmpffta[isamp].x << " " << tmpffta[isamp].y << " " << tmpfftb[isamp].x << " " << tmpfftb[isamp].y << endl;
    //                 }*/
    
    //                 fftfilea.write(reinterpret_cast<char*>(tmpffta), fftedsize * sizeof(hipfftComplex));
    //                 fftfileb.write(reinterpret_cast<char*>(tmpfftb), fftedsize * sizeof(hipfftComplex));
    
    //                 interend = std::chrono::steady_clock::now();
    //                 runtimes.intertime += std::chrono::duration<float>(interend - interstart).count();           
    
    //             }FilHead filhead;
    //     ReadFilterbankHeader(config, filhead);
        
    //     if (scaling) {
    //         filhead.nbits = 8;
    //     }
    
    //     // TODO: This will be wrong for R2C FFT
    //     filhead.tsamp = 1.0 / (2.0 * filhead.foff) * 2 * FFTUSE * TIMEAVG;
    //     // TODO: Make sure it is the middle of the top frequency channel
    //     filhead.fch1 = (filhead.fch1 + filhead.foff / 2.0f) * 1e-06;
    //     filhead.nchans = FFTUSE;
    //     filhead.foff = -1.0 * filhead.foff / FFTUSE * 1e-06 ;
    
    //     filhead.fch1 = filhead.fch1 + filhead.foff / 2.0;
    
    //     if (DEBUG) {
    //         cout << "Some header info:\n"
    //                 << "Raw file: " << filhead.rawfile << endl
    //                 << "Source name: " << filhead.source << endl
    //                 << "Azimuth: " << filhead.az << endl
    //                 << "Zenith angle: " << filhead.za << endl
    //                 << "Declination: " << filhead.dec << endl
    //                 << "Right ascension: " << filhead.ra << endl
    //                 << "Top channel frequency: " << filhead.fch1 << endl
    //                 << "Channel bandwidth: " << filhead.foff << endl
    //                 << "Number of channels: " << filhead.nchans << endl
    //                 << "Sampling time: " << filhead.tsamp << endl
    //                 << "Bits per sample: " << filhead.nbits << endl;
    //     }
    
    //     // TODO: Make sure there are correct values for bandwidth and sampling time in the header after taking averaging into account
    
    //     ifstream filepola(inpola.c_str(), ifstream::in | ifstream::binary);
    //     ifstream filepolb(inpolb.c_str(), ifstream::in | ifstream::binary);
    //     ofstream filfile(outfil.c_str(), ofstream::out | ofstream::binary);
    
    //     if (!filepola || !filepolb) {
    //     if (!filepola) {
    //             cout << "Could not open file " << inpola << endl;
    //         }
    //         if (!filepolb) {
    //             cout << "Could not open file " << inpolb << endl;
    //         }
    //         exit(EXIT_FAILURE);
    //     }
    //     // TODO: Can save the filterbank header straight away, after the first header is read
    //     unsigned char vdifheadpola[32];
    //     unsigned char vdifheadpolb[32];
    //     filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //     filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    
    //     filepola.seekg(0, filepola.end);
    //     long long filelengtha = filepola.tellg();
    //     filepola.seekg(0, filepola.beg);
    
    //     filepolb.seekg(0, filepolb.end);
    //     long long filelengthb = filepolb.tellg();
    //     filepolb.seekg(0, filepolb.beg);
    
    //     unsigned int startframe;
    //     unsigned int startsecond;
    
    //     cout << filelengtha << " " << filelengthb << endl;
    
    //     startframe = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));	// frame number in this second
    //     startsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    
    //     if (DEBUG) {
    //         cout << "Starting time: " << startsecond << ":" << startframe << endl;
    //     }
    
    //     // NOTE: Need to read headers in
    //     unsigned int toread = NACCUMULATE * 8000;
    //     // NOTE: No more headers after unpacking
    //     unsigned int unpackedsize = NACCUMULATE * VDIFSIZE * UNPACKFACTOR;
    //     unsigned int fftedsize = unpackedsize / (2 * FFTUSE) * FFTOUT;
    //     unsigned int powersize = unpackedsize / (2 * FFTUSE) * FFTUSE / TIMEAVG;
    
    //     hipfftHandle fftplan;
    //     int fftsizes[1];
    //     fftsizes[0] = 2 * FFTUSE;
    //     int fftbatchsize = unpackedsize / fftsizes[0];
    //     cout << fftbatchsize << endl;
    //     cufftCheckError(hipfftPlanMany(&fftplan, 1, fftsizes, NULL, 1, FFTUSE, NULL, 1, FFTUSE, HIPFFT_R2C, fftbatchsize));
    
    //     unsigned char *tmppola = new unsigned char[toread];
    //     unsigned char *tmppolb = new unsigned char[toread];
    
    //     unsigned char *devpola;
    //     unsigned char *devpolb;
    //     unsigned char **datapol = new unsigned char*[NPOL];
    //     unsigned char **devpol;
    //     float **unpacked = new float*[NPOL];
    //     float **devunpacked;
    //     hipfftComplex **ffted = new hipfftComplex*[NPOL];
    //     hipfftComplex **devffted;
        
    //     unsigned char *devpower;
    //     unsigned char *tmppower = new unsigned char[powersize * filhead.nbits / 8];
    
    //     if (GPURUN) {
    //         cudaCheckError(hipMalloc((void**)&devpola, toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&devpolb, toread * sizeof(unsigned char)));
    
    //         cudaCheckError(hipMalloc((void**)&devpol, NPOL * sizeof(unsigned char*)));
    //         cudaCheckError(hipMalloc((void**)&datapol[0], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMalloc((void**)&datapol[1], toread * sizeof(unsigned char)));
    //         cudaCheckError(hipMemcpy(devpol, datapol, NPOL * sizeof(unsigned char*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devunpacked, NPOL * sizeof(float*)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[0], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMalloc((void**)&unpacked[1], unpackedsize * sizeof(float)));
    //         cudaCheckError(hipMemcpy(devunpacked, unpacked, NPOL * sizeof(float*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devffted, NPOL * sizeof(hipfftComplex*)));
    //         cudaCheckError(hipMalloc((void**)&ffted[0], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMalloc((void**)&ffted[1], fftedsize * sizeof(hipfftComplex)));
    //         cudaCheckError(hipMemcpy(devffted, ffted, NPOL * sizeof(hipfftComplex*), hipMemcpyHostToDevice));
    
    //         cudaCheckError(hipMalloc((void**)&devpower, powersize * (filhead.nbits / 8)));
    //     }
    
    //     vector<std::pair<FrameInfo, FrameInfo>> vdifframes;
    
    //     FrameInfo tmpframea, tmpframeb;
    //     int refsecond;
    //     int frameno;
    //     int epoch;
    
    //     WriteFilterbankHeader(filfile, filhead);
       
    //     Timing runtimes;
    //     runtimes.readtime = 0.0f;
    //     runtimes.scaletime = 0.0f;
    //     runtimes.filtime = 0.0f;
    //     runtimes.savetime = 0.0f;
    //     runtimes.totaltime = 0.0f;
    //     runtimes.intertime = 0.0f;
    
    //     std::chrono::time_point<std::chrono::steady_clock> readstart, readend, scalestart, scaleend, filstart, filend, savestart, saveend, interstart, interend;
    
    //     float *tmpunpackeda = new float[unpackedsize];
    //     float *tmpunpackedb = new float[unpackedsize];
    //     hipfftComplex *tmpffta = new hipfftComplex[fftedsize];
    //     hipfftComplex *tmpfftb = new hipfftComplex[fftedsize];
    
    //     bool saved = false;
    
    //     //float *dmeans;
    //     //float *dstdevs;
    //     //cudaCheckError(hipMalloc((void**)&dmeans, FFTUSE * sizeof(float)));
    //     //cudaCheckError(hipMalloc((void**)&dstdevs, FFTUSE * sizeof(float)));
    
    //     thrust::device_vector<float> dmeans, dstdevs;
    //     dmeans.resize(FFTUSE);
    //     dstdevs.resize(FFTUSE);
    //     thrust::fill(dmeans.begin(), dmeans.end(), 0.0f);
    //     thrust::fill(dstdevs.begin(), dstdevs.end(), 0.0f);
    //     float *pdmeans = thrust::raw_pointer_cast(dmeans.data());
    //     float *pdstdevs = thrust::raw_pointer_cast(dstdevs.data());    
    
    //     cout << "Size of the device vectors: " << dmeans.size() << " " << dstdevs.size() << endl;
    
    //     scalestart = std::chrono::steady_clock::now();
    
    //     // NOTE: Use first 5 accumulates of data to obtain scaling factors
    //     if (scaling) {
    
    //         size_t divfactors = 5 * powersize / FFTUSE;
    //         thrust::device_vector<float> dfactors; 
    //         dfactors.resize(divfactors + 1);
    //         thrust::sequence(dfactors.begin(), dfactors.end());
    //         thrust::transform(dfactors.begin(), dfactors.end(), dfactors.begin(), FactorFunctor());
    //         float *pdfactors = thrust::raw_pointer_cast(dfactors.data());
    
    //         //float *dfactors;
    //         //size_t divfactors = 5 * powersize / FFTUSE;
    //         //cudaCheckError(hipMalloc((void**)&dfactors, divfactors * sizeof(float)));
    //         //int scalethreads = 1024;
    //         //int scaleblocks = (divfactors - 1) / scalethreads + 1;
    //         //cout << "Div factors blocks: " << scaleblocks << " and threads: " << scalethreads << endl;
    //         //InitDivFactors<<<scaleblocks, scalethreads>>>(dfactors, divfactors);
    //         //cudaCheckError(hipDeviceSynchronize());
    //         //cudaCheckError(hipGetLastError());
    //         size_t processed = 0;
    
    //         float *tmpdpower;
    //         cudaCheckError(hipMalloc((void**)&tmpdpower, powersize * sizeof(float)));
    
    //     while((filepola.tellg() < (5 * NACCUMULATE * 8032)) && (filepolb.tellg() < (5 * NACCUMULATE * 8032))) {
    //             for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //                 filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //                 filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //                 filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //                 filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             }
    
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    //             DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, tmpdpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             GetScalingFactorsKernel<<<1, FFTUSE, 0, 0>>>(tmpdpower, pdmeans, pdstdevs, pdfactors, processed);
    //             processed += (powersize / FFTUSE);
    //             cudaCheckError(hipDeviceSynchronize());
    //         }
    
    //         //float *hmeans = new float[FFTUSE];
    //         //float *hstdevs = new float[FFTUSE];
    
    //         //cudaCheckError(hipMemcpy(hmeans, dmeans, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    //         //cudaCheckError(hipMemcpy(hstdevs, dstdevs, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
    
    //         thrust::host_vector<float> hmeans = dmeans;
    //         thrust::host_vector<float> hstdevs = dstdevs;
    
    //         std::ofstream statsfile("mean_stdev.dat");
    
    //         cout << "Size of host vector:" << hmeans.size() << endl;
     
    //         if (statsfile) {
    //             for (int ichan = 0; ichan < hmeans.size(); ++ichan) {
    //                 statsfile << hmeans[ichan] << " " << hstdevs[ichan] << endl;
    //             }
    //         } else {
    //             cerr << "Could not open the stats file" << endl;
    //         }
    
    //         statsfile.close();
    
    //         hipFree(tmpdpower);
             
    //     }
    
    //     scaleend = std::chrono::steady_clock::now();
    
    //     runtimes.scaletime = std::chrono::duration<float>(scaleend - scalestart).count();
    
    //     filepola.seekg(0, filepola.beg);
    //     filepolb.seekg(0, filepolb.beg);
    
    //     std::ofstream unpackedfilea ((outfil + ".unp0").c_str(), std::ios_base::binary);
    //     std::ofstream unpackedfileb ((outfil + ".unp1").c_str(), std::ios_base::binary);
    //     std::ofstream fftfilea ((outfil + ".fft0").c_str(), std::ios_base::binary);
    //     std::ofstream fftfileb ((outfil + ".fft1").c_str(), std::ios_base::binary);
    
    //     while((filepola.tellg() < (filelengtha - NACCUMULATE * 8000)) && (filepolb.tellg() < (filelengthb - NACCUMULATE * 8000))) {
    //         //cout << filepola.tellg() << endl;
    //         // NOTE: This implementation
    //         for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
    //         readstart = std::chrono::steady_clock::now();
    //             filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    //             filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
    //             filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
    //             filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
    //             readend = std::chrono::steady_clock::now();
    //             runtimes.readtime += std::chrono::duration<float>(readend - readstart).count();
    
    //             refsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));
    //             epoch = (unsigned int)(vdifheadpola[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframea.frameno = frameno;
    //             tmpframea.refsecond = refsecond;
    //             tmpframea.refepoch = epoch;
    
    //             refsecond = (unsigned int)(vdifheadpolb[0] | (vdifheadpolb[1] << 8) | (vdifheadpolb[2] << 16) | ((vdifheadpolb[3] & 0x3f) << 24));
    //             frameno = (unsigned int)(vdifheadpolb[4] | (vdifheadpolb[5] << 8) | (vdifheadpolb[6] << 16));
    //             epoch = (unsigned int)(vdifheadpolb[7] & 0x3f);
    // //            frameno += (refsecond - startsecond) * 4000;
    
    //             tmpframeb.frameno = frameno;
    //             tmpframeb.refsecond = refsecond;
    //             tmpframeb.refepoch = epoch;
    
    //             vdifframes.push_back(std::make_pair(tmpframea, tmpframeb));
    
    //             // NOTE: Can use subtract startframe to put frame count at 0 and use that to save into the buffer
    
    //         }
     
    //        if (GPURUN) {
    //             filstart = std::chrono::steady_clock::now();
    //             cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    //             cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    //             UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
    //             for (int ipol = 0; ipol < NPOL; ++ipol) {
    //                 cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
    //             }
    
    //             if (filhead.nbits == 8) {
    //                 DetectScaleKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<unsigned char*>(devpower), pdmeans, pdstdevs);
    //             } else if (filhead.nbits == 32) {
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             } else {
    //                 cerr << "Unsupported option! Will use float!" << endl;
    //                 DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
    //             }
    
    //             //PowerKernel<<<25, FFTUSE, 0, 0>>>(devffted, devpower);
    //             cudaCheckError(hipDeviceSynchronize());
    //             cudaCheckError(hipMemcpy(tmppower, devpower, powersize * filhead.nbits / 8, hipMemcpyDeviceToHost));
                
    //             if (!saved) {
    //                 std::ofstream unpackedfile("unpacked.dat");
    //         cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
    //         for (int isamp = 0; isamp < 2 * 8000 * 4; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }
    //                 unpackedfile.close();
    //                 saved = true;
    //             }
    
    //             filend = std::chrono::steady_clock::now();
    //             runtimes.filtime += std::chrono::duration<float>(filend - filstart).count();
                
    //             savestart = std::chrono::steady_clock::now(); 
    //             filfile.write(reinterpret_cast<char*>(tmppower), powersize * filhead.nbits / 8);
    //             saveend = std::chrono::steady_clock::now();
    //             runtimes.savetime += std::chrono::duration<float>(saveend - savestart).count();
    
       
    
    //             if (saveinter) {
    
    //                 interstart = std::chrono::steady_clock::now();
    
    //                 cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //         cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < unpackedsize; ++isamp) {
    //                     unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
    //                 }*/
    
    //                 unpackedfilea.write(reinterpret_cast<char*>(tmpunpackeda), unpackedsize * sizeof(float));
    //                 unpackedfileb.write(reinterpret_cast<char*>(tmpunpackedb), unpackedsize * sizeof(float));
    
    //                 cudaCheckError(hipMemcpy(tmpffta, ffted[0], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 cudaCheckError(hipMemcpy(tmpfftb, ffted[1], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    //                 /*for (int isamp = 0; isamp < fftedsize; ++isamp) {
    //                     fftfile << tmpffta[isamp].x << " " << tmpffta[isamp].y << " " << tmpfftb[isamp].x << " " << tmpfftb[isamp].y << endl;
    //                 }*/
    
    //                 fftfilea.write(reinterpret_cast<char*>(tmpffta), fftedsize * sizeof(hipfftComplex));
    //                 fftfileb.write(reinterpret_cast<char*>(tmpfftb), fftedsize * sizeof(hipfftComplex));
    
    //                 interend = std::chrono::steady_clock::now();
    //                 runtimes.intertime += std::chrono::duration<float>(interend - interstart).count();           
    
    //             }
    
    //         }
    //         cout << "Completed " << std::fixed << std::setprecision(2) << (float)filepola.tellg() / (float)(filelengtha - 1.0) * 100.0f << "%\r";
    //         cout.flush();
    //     }
    
    //     cout << endl;
    //     filfile.close();
    //     unpackedfilea.close();
    //     unpackedfileb.close();
    //     fftfilea.close();
    //     fftfileb.close();
    
    //     runtimes.totaltime = runtimes.readtime + runtimes.scaletime + runtimes.filtime + runtimes.savetime + runtimes.intertime;
    
    //     cout << "Total execution time: " << runtimes.totaltime << "s\n";
    //     cout << "\tScaling factors: " << runtimes.scaletime << "s\n";
    //     cout << "\tFile read: " << runtimes.readtime << "s\n";
    //     cout << "\tFilterbanking: " << runtimes.filtime << "s\n";
    //     cout << "\tFile write: " << runtimes.savetime << "s\n";
    //     if (saveinter) {
    //         cout << "\tIntermediate write: " << runtimes.intertime << "s\n";
    //     }
    
    //     if (DEBUG) {
    //         std::ofstream outframes("dataframes.dat");
    
    //         outframes << "Ref Epoch A\tRef second A\tRef frame A\tRef Epoch B\tRef second B\tRef frame b\n";
    //         for (auto iframe = vdifframes.begin(); iframe != vdifframes.end(); ++iframe) {
    //             outframes << iframe->first.refepoch << "\t" << iframe->first.refsecond << "\t" << iframe->first.frameno << "\t"
    //             << iframe->second.refepoch << "\t" << iframe->second.refsecond << "\t" << iframe->second.frameno << endl;
    //         }
    
    //         outframes.close();
    //     }


    // }

    return 0;
}
